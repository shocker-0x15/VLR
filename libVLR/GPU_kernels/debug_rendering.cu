#include "hip/hip_runtime.h"
﻿#include "../shared/renderer_common.h"

namespace vlr {
    using namespace shared;

    // for debug rendering
    CUDA_DEVICE_FUNCTION TripletSpectrum debugRenderingAttributeToSpectrum(const SurfacePoint &surfPt, DebugRenderingAttribute attribute) {
        TripletSpectrum value;

        switch (attribute) {
        case DebugRenderingAttribute::GeometricNormal:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.z));
            break;
        case DebugRenderingAttribute::ShadingTangent:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.z));
            break;
        case DebugRenderingAttribute::ShadingBitangent:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.z));
            break;
        case DebugRenderingAttribute::ShadingNormal:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.z));
            break;
        case DebugRenderingAttribute::TextureCoordinates:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          surfPt.texCoord.u - ::vlr::floor(surfPt.texCoord.u),
                                          surfPt.texCoord.v - ::vlr::floor(surfPt.texCoord.v),
                                          0.0f);
            break;
        case DebugRenderingAttribute::GeometricVsShadingNormal: {
            float sim = dot(surfPt.geometricNormal, surfPt.shadingFrame.z);
            bool opposite = sim < 0.0f;
            sim = std::fabs(sim);
            const float coeff = 5.0f;
            float sValue = 0.5f + coeff * (sim - 1);
            sValue = clamp(sValue, 0.0f, 1.0f);
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65, sValue, opposite ? 0 : sValue, opposite ? 0 : sValue);
            break;
        }
        case DebugRenderingAttribute::ShadingFrameLengths:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.x.length() - 1), 0.0f, 1.0f),
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.y.length() - 1), 0.0f, 1.0f),
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.z.length() - 1), 0.0f, 1.0f));
            break;
        case DebugRenderingAttribute::ShadingFrameOrthogonality:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.x, surfPt.shadingFrame.y), 0.0f, 1.0f),
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.y, surfPt.shadingFrame.z), 0.0f, 1.0f),
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.z, surfPt.shadingFrame.x), 0.0f, 1.0f));
            break;
        default:
            break;
        }

        return value;
    }




    // Common Any Hit Program for All Primitive Types and Materials
    CUDA_DEVICE_KERNEL void RT_AH_NAME(debugRenderingAnyHitWithAlpha)() {
        KernelRNG rng;
        WavelengthSamples wls;
        DebugPayloadSignature::get(&rng, &wls, nullptr);

        float alpha = getAlpha(wls);

        // Stochastic Alpha Test
        if (rng.getFloat0cTo1o() >= alpha)
            optixIgnoreIntersection();

        DebugPayloadSignature::set(&rng, nullptr, nullptr);
    }



    // Common Closest Hit Program for All Primitive Types and Materials
    CUDA_DEVICE_KERNEL void RT_CH_NAME(debugRenderingClosestHit)() {
        const auto hp = HitPointParameter::get();

        WavelengthSamples wls;
        DebugPayloadSignature::get(nullptr, &wls, nullptr);

        SurfacePoint surfPt;
        float hypAreaPDF;
        calcSurfacePoint(hp, wls, &surfPt, &hypAreaPDF);

        //if (!surfPt.shadingFrame.x.allFinite() ||
        //    !surfPt.shadingFrame.y.allFinite() ||
        //    !surfPt.shadingFrame.z.allFinite())
        //    vlrprintf("(%g, %g, %g), (%g, %g, %g), (%g, %g, %g)\n",
        //              surfPt.shadingFrame.x.x, surfPt.shadingFrame.x.y, surfPt.shadingFrame.x.z,
        //              surfPt.shadingFrame.y.x, surfPt.shadingFrame.y.y, surfPt.shadingFrame.y.z,
        //              surfPt.shadingFrame.z.x, surfPt.shadingFrame.z.y, surfPt.shadingFrame.z.z);

        SampledSpectrum value;
        if (plp.debugRenderingAttribute == DebugRenderingAttribute::BaseColor) {
            const SurfaceMaterialDescriptor matDesc = plp.materialDescriptorBuffer[hp.sbtr->geomInst.materialIndex];
            BSDF bsdf(matDesc, surfPt, wls);

            const BSDFProcedureSet procSet = plp.bsdfProcedureSetBuffer[matDesc.bsdfProcedureSetIndex];
            auto progGetBaseColor = static_cast<ProgSigBSDFGetBaseColor>(procSet.progGetBaseColor);

            TripletSpectrum whitePoint = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                                               1, 1, 1);
            value = progGetBaseColor(reinterpret_cast<const uint32_t*>(&bsdf)) * whitePoint.evaluate(wls);
        }
        else {
            value = debugRenderingAttributeToSpectrum(surfPt, plp.debugRenderingAttribute).evaluate(wls);
        }

        DebugPayloadSignature::set(nullptr, nullptr, &value);
    }



    // JP: 本当は無限大の球のIntersection/Bounding Box Programを使用して環境光に関する処理もClosest Hit Programで統一的に行いたい。
    //     が、OptiXのBVHビルダーがLBVHベースなので無限大のAABBを生成するのは危険。
    //     仕方なくMiss Programで環境光を処理する。
    CUDA_DEVICE_KERNEL void RT_MS_NAME(debugRenderingMiss)() {
        WavelengthSamples wls;
        DebugPayloadSignature::get(nullptr, &wls, nullptr);

        const Instance &inst = plp.instBuffer[plp.envLightInstIndex];
        //const GeometryInstance &geomInst = plp.geomInstBuffer[inst.geomInstIndices[0]];

        Vector3D direction = asVector3D(optixGetWorldRayDirection());
        float phi, theta;
        direction.toPolarYUp(&theta, &phi);

        float sinPhi, cosPhi;
        ::vlr::sincos(phi, &sinPhi, &cosPhi);
        Vector3D texCoord0Dir = normalize(Vector3D(-cosPhi, 0.0f, -sinPhi));
        ReferenceFrame shadingFrame;
        shadingFrame.x = texCoord0Dir;
        shadingFrame.z = -direction;
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint surfPt;
        surfPt.position = Point3D(direction.x, direction.y, direction.z);
        surfPt.shadingFrame = shadingFrame;
        surfPt.isPoint = false;
        surfPt.atInfinity = true;

        surfPt.geometricNormal = -direction;
        surfPt.u = phi;
        surfPt.v = theta;
        phi += inst.rotationPhi;
        phi = phi - ::vlr::floor(phi / (2 * VLR_M_PI)) * 2 * VLR_M_PI;
        surfPt.texCoord = TexCoord2D(phi / (2 * VLR_M_PI), theta / VLR_M_PI);

        SampledSpectrum value;
        if (plp.debugRenderingAttribute == DebugRenderingAttribute::BaseColor)
            value = SampledSpectrum::Zero();
        else
            value = debugRenderingAttributeToSpectrum(surfPt, plp.debugRenderingAttribute).evaluate(wls);

        DebugPayloadSignature::set(nullptr, nullptr, &value);
    }



    // Common Ray Generation Program for All Camera Types
    CUDA_DEVICE_KERNEL void RT_RG_NAME(debugRenderingRayGeneration)() {
        uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

        KernelRNG rng = plp.rngBuffer.read(launchIndex);

        float2 p = make_float2(launchIndex.x + rng.getFloat0cTo1o(),
                               launchIndex.y + rng.getFloat0cTo1o());

        float selectWLPDF;
        WavelengthSamples wls = WavelengthSamples::createWithEqualOffsets(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), &selectWLPDF);

        ProgSigSampleLensPosition sampleLensPosition(plp.progSampleLensPosition);
        ProgSigSampleIDF sampleIDF(plp.progSampleIDF);

        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        SampledSpectrum We0 = sampleLensPosition(wls, We0Sample, &We0Result);

        IDFSample We1Sample(p.x / plp.imageSize.x, p.y / plp.imageSize.y);
        IDFQueryResult We1Result;
        SampledSpectrum We1 = sampleIDF(We0Result.surfPt, wls, We1Sample, &We1Result);

        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        SampledSpectrum alpha = (We0 * We1) * (We0Result.surfPt.calcCosTerm(rayDir) / (We0Result.areaPDF * We1Result.dirPDF * selectWLPDF));

        SampledSpectrum value;
        optixu::trace<DebugPayloadSignature>(
            plp.topGroup, asOptiXType(We0Result.surfPt.position), asOptiXType(rayDir), 0.0f, FLT_MAX, 0.0f,
            0xFF, OPTIX_RAY_FLAG_NONE,
            DebugRayType::Primary, MaxNumRayTypes, DebugRayType::Primary,
            rng, wls, value);

        plp.rngBuffer.write(launchIndex, rng);

        if (!value.allFinite()) {
            vlrprintf("Pass %u, (%u, %u): Not a finite value.\n", plp.numAccumFrames, launchIndex.x, launchIndex.y);
            return;
        }

        if (plp.numAccumFrames == 1)
            plp.accumBuffer[launchIndex].reset();
        plp.accumBuffer[launchIndex].add(wls, value / selectWLPDF);
    }
}
