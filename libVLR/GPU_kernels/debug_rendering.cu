#include "hip/hip_runtime.h"
﻿#include "light_transport_common.cuh"

namespace VLR {
    struct DebugRenderingPayload {
        KernelRNG rng;
        WavelengthSamples wls;
        SampledSpectrum value;
    };

#define DebugPayloadSignature DebugRenderingPayload*



    // for debug rendering
    CUDA_DEVICE_FUNCTION TripletSpectrum debugRenderingAttributeToSpectrum(const SurfacePoint &surfPt, DebugRenderingAttribute attribute) {
        TripletSpectrum value;

        switch (attribute) {
        case DebugRenderingAttribute::GeometricNormal:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.geometricNormal.z));
            break;
        case DebugRenderingAttribute::ShadingTangent:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.x.z));
            break;
        case DebugRenderingAttribute::ShadingBitangent:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.y.z));
            break;
        case DebugRenderingAttribute::ShadingNormal:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.x),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.y),
                                          std::fmax(0.0f, 0.5f + 0.5f * surfPt.shadingFrame.z.z));
            break;
        case DebugRenderingAttribute::TextureCoordinates:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          surfPt.texCoord.u - std::floor(surfPt.texCoord.u),
                                          surfPt.texCoord.v - std::floor(surfPt.texCoord.v),
                                          0.0f);
            break;
        case DebugRenderingAttribute::GeometricVsShadingNormal: {
            float sim = dot(surfPt.geometricNormal, surfPt.shadingFrame.z);
            bool opposite = sim < 0.0f;
            sim = std::fabs(sim);
            const float coeff = 5.0f;
            float sValue = 0.5f + coeff * (sim - 1);
            sValue = clamp(sValue, 0.0f, 1.0f);
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65, sValue, opposite ? 0 : sValue, opposite ? 0 : sValue);
            break;
        }
        case DebugRenderingAttribute::ShadingFrameLengths:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.x.length() - 1), 0.0f, 1.0f),
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.y.length() - 1), 0.0f, 1.0f),
                                          clamp(0.5f + 10 * (surfPt.shadingFrame.z.length() - 1), 0.0f, 1.0f));
            break;
        case DebugRenderingAttribute::ShadingFrameOrthogonality:
            value = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.x, surfPt.shadingFrame.y), 0.0f, 1.0f),
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.y, surfPt.shadingFrame.z), 0.0f, 1.0f),
                                          clamp(0.5f + 100 * dot(surfPt.shadingFrame.z, surfPt.shadingFrame.x), 0.0f, 1.0f));
            break;
        default:
            break;
        }

        return value;
    }




    // Common Any Hit Program for All Primitive Types and Materials
    CUDA_DEVICE_KERNEL void RT_AH_NAME(debugRenderingAnyHitWithAlpha)() {
        DebugRenderingPayload* payload;
        optixu::getPayloads<DebugPayloadSignature>(&payload);

        float alpha = getAlpha(payload->wls);

        // Stochastic Alpha Test
        if (payload->rng.getFloat0cTo1o() >= alpha)
            optixIgnoreIntersection();
    }



    // Common Closest Hit Program for All Primitive Types and Materials
    CUDA_DEVICE_KERNEL void RT_CH_NAME(debugRenderingClosestHit)() {
        const auto &sbtr = HitGroupSBTRecordData::get();

        DebugRenderingPayload* payload;
        optixu::getPayloads<DebugPayloadSignature>(&payload);

        WavelengthSamples &wls = payload->wls;

        SurfacePoint surfPt;
        float hypAreaPDF;
        calcSurfacePoint(wls, &surfPt, &hypAreaPDF);

        //if (!surfPt.shadingFrame.x.allFinite() || !surfPt.shadingFrame.y.allFinite() || !surfPt.shadingFrame.z.allFinite())
        //    vlrprintf("(%g, %g, %g), (%g, %g, %g), (%g, %g, %g)\n",
        //              surfPt.shadingFrame.x.x, surfPt.shadingFrame.x.y, surfPt.shadingFrame.x.z,
        //              surfPt.shadingFrame.y.x, surfPt.shadingFrame.y.y, surfPt.shadingFrame.y.z,
        //              surfPt.shadingFrame.z.x, surfPt.shadingFrame.z.y, surfPt.shadingFrame.z.z);

        if (plp.debugRenderingAttribute == DebugRenderingAttribute::BaseColor) {
            const SurfaceMaterialDescriptor matDesc = plp.materialDescriptorBuffer[sbtr.geomInst.materialIndex];
            BSDF bsdf(matDesc, surfPt, wls);

            const BSDFProcedureSet procSet = plp.bsdfProcedureSetBuffer[matDesc.bsdfProcedureSetIndex];
            auto progGetBaseColor = (ProgSigBSDFGetBaseColor)procSet.progGetBaseColor;

            payload->value = progGetBaseColor((const uint32_t*)&bsdf);
        }
        else {
            payload->value = debugRenderingAttributeToSpectrum(surfPt, plp.debugRenderingAttribute).evaluate(wls);
        }
    }



    // JP: 本当は無限大の球のIntersection/Bounding Box Programを使用して環境光に関する処理もClosest Hit Programで統一的に行いたい。
    //     が、OptiXのBVHビルダーがLBVHベースなので無限大のAABBを生成するのは危険。
    //     仕方なくMiss Programで環境光を処理する。
    CUDA_DEVICE_KERNEL void RT_MS_NAME(debugRenderingMiss)() {
        DebugRenderingPayload* payload;
        optixu::getPayloads<DebugPayloadSignature>(&payload);

        WavelengthSamples &wls = payload->wls;

        Vector3D direction = asVector3D(optixGetWorldRayDirection());
        float phi, theta;
        direction.toPolarYUp(&theta, &phi);

        float sinPhi, cosPhi;
        VLR::sincos(phi, &sinPhi, &cosPhi);
        Vector3D texCoord0Dir = normalize(Vector3D(-cosPhi, 0.0f, -sinPhi));
        ReferenceFrame shadingFrame;
        shadingFrame.x = texCoord0Dir;
        shadingFrame.z = -direction;
        shadingFrame.y = cross(shadingFrame.z, shadingFrame.x);

        SurfacePoint surfPt;
        surfPt.position = Point3D(direction.x, direction.y, direction.z);
        surfPt.shadingFrame = shadingFrame;
        surfPt.isPoint = false;
        surfPt.atInfinity = true;

        surfPt.geometricNormal = -direction;
        surfPt.u = phi;
        surfPt.v = theta;
        phi += plp.envLightDescriptor.rotationPhi;
        phi = phi - std::floor(phi / (2 * VLR_M_PI)) * 2 * VLR_M_PI;
        surfPt.texCoord = TexCoord2D(phi / (2 * VLR_M_PI), theta / VLR_M_PI);

        if (plp.debugRenderingAttribute == DebugRenderingAttribute::BaseColor) {
            payload->value = SampledSpectrum::Zero();
        }
        else {
            payload->value = debugRenderingAttributeToSpectrum(surfPt, plp.debugRenderingAttribute).evaluate(wls);
        }
    }



    // Common Ray Generation Program for All Camera Types
    CUDA_DEVICE_KERNEL void RT_RG_NAME(debugRenderingRayGeneration)() {
        uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

        KernelRNG rng = plp.rngBuffer[launchIndex];

        float2 p = make_float2(launchIndex.x + rng.getFloat0cTo1o(),
                               launchIndex.y + rng.getFloat0cTo1o());

        float selectWLPDF;
        WavelengthSamples wls = WavelengthSamples::createWithEqualOffsets(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), &selectWLPDF);

        ProgSigSampleLensPosition sampleLensPosition(plp.progSampleLensPosition);
        ProgSigSampleIDF sampleIDF(plp.progSampleIDF);

        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        SampledSpectrum We0 = sampleLensPosition(wls, We0Sample, &We0Result);

        IDFSample We1Sample(p.x / plp.imageSize.x, p.y / plp.imageSize.y);
        IDFQueryResult We1Result;
        SampledSpectrum We1 = sampleIDF(We0Result.surfPt, wls, We1Sample, &We1Result);

        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        SampledSpectrum alpha = (We0 * We1) * (We0Result.surfPt.calcCosTerm(rayDir) / (We0Result.areaPDF * We1Result.dirPDF * selectWLPDF));

        DebugRenderingPayload payload;
        payload.rng = rng;
        payload.wls = wls;
        DebugRenderingPayload* payloadPtr = &payload;
        optixu::trace<DebugPayloadSignature>(
            plp.topGroup, asOptiXType(We0Result.surfPt.position), asOptiXType(rayDir), 0.0f, FLT_MAX, 0.0f,
            0xFF, OPTIX_RAY_FLAG_NONE,
            RayType::DebugPrimary, RayType::NumTypes, RayType::DebugPrimary,
            payloadPtr);

        plp.rngBuffer[launchIndex] = payload.rng;

        if (!payload.value.allFinite()) {
            vlrprintf("Pass %u, (%u, %u): Not a finite value.\n", plp.numAccumFrames, launchIndex.x, launchIndex.y);
            return;
        }

        if (plp.numAccumFrames == 1)
            plp.outputBuffer[launchIndex].reset();
        plp.outputBuffer[launchIndex].add(wls, payload.value / selectWLPDF);
    }
}
