﻿#include "../shared/kernel_common.h"

namespace vlr {
    using namespace shared;

    RT_CALLABLE_PROGRAM void RT_DC_NAME(decodeHitPointForTriangle)(
        const HitPointParameter &param, SurfacePoint* surfPt, float* hypAreaPDF) {
        const GeometryInstance &geomInst = param.sbtr->geomInst;

        const Triangle &triangle = geomInst.asTriMesh.triangleBuffer[param.primIndex];
        const Vertex &v0 = geomInst.asTriMesh.vertexBuffer[triangle.index0];
        const Vertex &v1 = geomInst.asTriMesh.vertexBuffer[triangle.index1];
        const Vertex &v2 = geomInst.asTriMesh.vertexBuffer[triangle.index2];

        Vector3D e1 = v1.position - v0.position;
        Vector3D e2 = v2.position - v0.position;
        Normal3D geometricNormal = cross(e1, e2);
        float area = geometricNormal.length() / 2; // TODO: スケーリングの考慮。
        geometricNormal /= 2 * area;

        // JP: プログラムがこの点を光源としてサンプルする場合の面積に関する(仮想的な)PDFを求める。
        // EN: calculate a hypothetical area PDF value in the case where the program sample this point as light.
        float probLightPrim = area / geomInst.asTriMesh.primDistribution.integral();
        *hypAreaPDF = probLightPrim / area;

        float b0 = 1 - param.b1 - param.b2, b1 = param.b1, b2 = param.b2;
        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = normalize(b0 * v0.normal + b1 * v1.normal + b2 * v2.normal);
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = tc0Direction - dotNT * shadingNormal;

        surfPt->position = position;
        surfPt->shadingFrame.x = tc0Direction;
        surfPt->shadingFrame.z = shadingNormal;
        surfPt->isPoint = false;
        surfPt->atInfinity = false;
        surfPt->geometricNormal = geometricNormal;
        surfPt->u = b0;
        surfPt->v = b1;
        surfPt->texCoord = texCoord;
    }



    RT_CALLABLE_PROGRAM void RT_DC_NAME(sampleTriangleMesh)(
        const Instance &inst, const GeometryInstance &geomInst,
        const SurfaceLightPosSample &sample, const Point3D &shadingPoint,
        SurfaceLightPosQueryResult* result) {
        (void)shadingPoint;

        float primProb;
        uint32_t primIdx = geomInst.asTriMesh.primDistribution.sample(sample.uElem, &primProb);
        //printf("%g, %u, %g\n", sample.uElem, primIdx, primProb);

        const Triangle &triangle = geomInst.asTriMesh.triangleBuffer[primIdx];
        const Vertex &v0 = geomInst.asTriMesh.vertexBuffer[triangle.index0];
        const Vertex &v1 = geomInst.asTriMesh.vertexBuffer[triangle.index1];
        const Vertex &v2 = geomInst.asTriMesh.vertexBuffer[triangle.index2];

        const StaticTransform &transform = inst.transform;

        Vector3D e1 = transform * (v1.position - v0.position);
        Vector3D e2 = transform * (v2.position - v0.position);
        Normal3D geometricNormal = cross(e1, e2);
        float area = geometricNormal.length() / 2;
        geometricNormal /= 2 * area;

        result->areaPDF = primProb / area;
        result->posType = DirectionType::Emission() | DirectionType::LowFreq();
        result->materialIndex = geomInst.materialIndex;

        float b0, b1, b2;
        uniformSampleTriangle(sample.uPos[0], sample.uPos[1], &b0, &b1);
        b2 = 1.0f - b0 - b1;

        Point3D position = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        Vector3D tc0Direction = b0 * v0.tc0Direction + b1 * v1.tc0Direction + b2 * v2.tc0Direction;
        TexCoord2D texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        position = transform * position;
        shadingNormal = normalize(transform * shadingNormal);
        tc0Direction = transform * tc0Direction;

        // JP: 法線と接線が直交することを保証する。
        //     直交性の消失は重心座標補間によっておこる？
        // EN: guarantee the orthogonality between the normal and tangent.
        //     Orthogonality break might be caused by barycentric interpolation?
        float dotNT = dot(shadingNormal, tc0Direction);
        tc0Direction = normalize(tc0Direction - dotNT * shadingNormal);

        SurfacePoint &surfPt = result->surfPt;

        surfPt.position = position;
        surfPt.shadingFrame = ReferenceFrame(tc0Direction, shadingNormal);
        surfPt.isPoint = false;
        surfPt.atInfinity = false;
        surfPt.geometricNormal = geometricNormal;
        surfPt.u = b0;
        surfPt.v = b1;
        surfPt.texCoord = texCoord;
    }
}
