#include "hip/hip_runtime.h"
﻿#define RT_PIPELINE_LAUNCH_PARAMETERS extern "C" __constant__
#include "../shared/kernel_common.h"

namespace vlr {
    using namespace shared;

    CUDA_DEVICE_KERNEL void resetAtomicAccumBuffer(
        DiscretizedSpectrum* atomicAccumBuffer,
        uint2 imageSize, uint32_t imageStrideInPixels) {
        uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                       blockDim.y * blockIdx.y + threadIdx.y);
        if (launchIndex.x >= imageSize.x || launchIndex.y >= imageSize.y)
            return;
        uint32_t linearIndex = launchIndex.y * imageStrideInPixels + launchIndex.x;
        atomicAccumBuffer[linearIndex] = DiscretizedSpectrum::Zero();
    }

    CUDA_DEVICE_KERNEL void accumulateFromAtomicAccumBuffer(
        const DiscretizedSpectrum* atomicAccumBuffer,
        optixu::BlockBuffer2D<SpectrumStorage, 0> accumBuffer,
        uint2 imageSize, uint32_t imageStrideInPixels, uint32_t reset) {
        uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                       blockDim.y * blockIdx.y + threadIdx.y);
        if (launchIndex.x >= imageSize.x || launchIndex.y >= imageSize.y)
            return;
        uint32_t linearIndex = launchIndex.y * imageStrideInPixels + launchIndex.x;
        const DiscretizedSpectrum &srcValue = atomicAccumBuffer[linearIndex];
        if (reset)
            accumBuffer[launchIndex].reset();
        accumBuffer[launchIndex].add(srcValue);
    }

    CUDA_DEVICE_KERNEL void copyBuffers(const optixu::BlockBuffer2D<SpectrumStorage, 0> accumBuffer,
                                        const DiscretizedSpectrum* accumAlbedoBuffer,
                                        const Normal3D* accumNormalBuffer,
                                        Quaternion invOrientation,
                                        uint2 imageSize, uint32_t imageStrideInPixels,
                                        uint32_t numAccumFrames,
                                        float4* linearColorBuffer,
                                        float4* linearAlbedoBuffer,
                                        float4* linearNormalBuffer) {
        uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                       blockDim.y * blockIdx.y + threadIdx.y);
        uint32_t linearIndex = launchIndex.y * imageStrideInPixels + launchIndex.x;

        if (launchIndex.x >= imageSize.x || launchIndex.y >= imageSize.y)
            return;

        float recNumAccums = 1.0f / numAccumFrames;
        const DiscretizedSpectrum &color = accumBuffer[launchIndex].getValue().result;
        float colorXYZ[3];
        color.toXYZ(colorXYZ);
        colorXYZ[0] *= recNumAccums;
        colorXYZ[1] *= recNumAccums;
        colorXYZ[2] *= recNumAccums;
        VLRAssert(colorXYZ[0] >= 0.0f && colorXYZ[1] >= 0.0f && colorXYZ[2] >= 0.0f,
                  "each value of color XYZ must not be negative.");
        float colorRGB[3];
        transformTristimulus(mat_XYZ_to_Rec709_D65, colorXYZ, colorRGB);
        // JP: 同じセンサーサイズ設定で解像度を変えるとピクセル辺りの積分値の大きさが変わって(=明るさが変わって)
        //     使いづらいのでキャンセルする補正項をかける。
        // EN: Changing the resolution while keeping the sensor size causes difference in the magnitude of
        //     per-pixel integrated value (i.e. difference in brightness), making it difficult to use,
        //     so multiply a correction factor.
        float resCorrection = imageSize.x * imageSize.y;
        colorRGB[0] *= resCorrection;
        colorRGB[1] *= resCorrection;
        colorRGB[2] *= resCorrection;

        const DiscretizedSpectrum &albedo = accumAlbedoBuffer[linearIndex];
        float albedoXYZ[3];
        albedo.toXYZ(albedoXYZ);
        albedoXYZ[0] *= recNumAccums;
        albedoXYZ[1] *= recNumAccums;
        albedoXYZ[2] *= recNumAccums;
        VLRAssert(albedoXYZ[0] >= 0.0f && albedoXYZ[1] >= 0.0f && albedoXYZ[2] >= 0.0f,
                  "Pixel %u, %u: (%g, %g, %g), each value of albedo XYZ must not be negative.",
                  launchIndex.x, launchIndex.y, albedoXYZ[0], albedoXYZ[1], albedoXYZ[2]);
        float albedoRGB[3];
        transformTristimulus(mat_XYZ_to_Rec709_D65, albedoXYZ, albedoRGB);

        Normal3D normal = accumNormalBuffer[linearIndex];
        normal = invOrientation.toMatrix3x3() * normal;
        normal.x *= -1;
        if (normal.x != 0 || normal.y != 0 || normal.z != 0)
            normal.normalize();

        linearColorBuffer[linearIndex] = make_float4(colorRGB[0], colorRGB[1], colorRGB[2], 1.0f);
        linearAlbedoBuffer[linearIndex] = make_float4(albedoRGB[0], albedoRGB[1], albedoRGB[2], 1.0f);
        linearNormalBuffer[linearIndex] = make_float4(normal.x, normal.y, normal.z, 1.0f);
    }

    CUDA_DEVICE_KERNEL void convertToRGB(const optixu::BlockBuffer2D<SpectrumStorage, 0> accumBuffer,
                                         const float4* linearDenoisedColorBuffer,
                                         const float4* linearAlbedoBuffer,
                                         const float4* linearNormalBuffer,
                                         bool useDenoiser, bool debugRender, DebugRenderingAttribute debugAttr,
                                         uint2 imageSize, uint32_t imageStrideInPixels, uint32_t numAccumFrames,
                                         optixu::NativeBlockBuffer2D<float4> outputBuffer) {
        uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                       blockDim.y * blockIdx.y + threadIdx.y);
        if (launchIndex.x >= imageSize.x || launchIndex.y >= imageSize.y)
            return;

        float RGB[3];
        if (debugRender &&
            (debugAttr == DebugRenderingAttribute::DenoiserAlbedo ||
             debugAttr == DebugRenderingAttribute::DenoiserNormal)) {
            uint32_t linearIndex = launchIndex.y * imageStrideInPixels + launchIndex.x;
            float4 value;
            switch (debugAttr) {
            case DebugRenderingAttribute::DenoiserAlbedo:
                value = linearAlbedoBuffer[linearIndex];
                break;
            case DebugRenderingAttribute::DenoiserNormal:
                value = linearNormalBuffer[linearIndex];
                value = make_float4(0.5f * value.x + 0.5f,
                                    0.5f * value.y + 0.5f,
                                    0.5f * value.z + 0.5f,
                                    value.w);
                break;
            }
            RGB[0] = value.x;
            RGB[1] = value.y;
            RGB[2] = value.z;
        }
        else {
            if (useDenoiser) {
                uint32_t linearIndex = launchIndex.y * imageStrideInPixels + launchIndex.x;
                float4 value = linearDenoisedColorBuffer[linearIndex];
                RGB[0] = value.x;
                RGB[1] = value.y;
                RGB[2] = value.z;
            }
            else {
                const DiscretizedSpectrum &spectrum = accumBuffer[launchIndex].getValue().result;
                float XYZ[3];
                spectrum.toXYZ(XYZ);
                float recNumAccums = 1.0f / numAccumFrames;
                XYZ[0] *= recNumAccums;
                XYZ[1] *= recNumAccums;
                XYZ[2] *= recNumAccums;
                VLRAssert(XYZ[0] >= 0.0f && XYZ[1] >= 0.0f && XYZ[2] >= 0.0f,
                          "each value of XYZ must not be negative (%g, %g, %g).",
                          XYZ[0], XYZ[1], XYZ[2]);
                transformTristimulus(mat_XYZ_to_Rec709_D65, XYZ, RGB);

                if (!debugRender) {
                    // JP: 同じセンサーサイズ設定で解像度を変えるとピクセル辺りの積分値の大きさが変わって(=明るさが変わって)
                    //     使いづらいのでキャンセルする補正項をかける。
                    // EN: Changing the resolution while keeping the sensor size causes difference in the magnitude of
                    //     per-pixel integrated value (i.e. difference in brightness), making it difficult to use,
                    //     so multiply a correction factor.
                    float resCorrection = imageSize.x * imageSize.y;
                    RGB[0] *= resCorrection;
                    RGB[1] *= resCorrection;
                    RGB[2] *= resCorrection;
                }
            }
        }

        outputBuffer.write(launchIndex, make_float4(RGB[0], RGB[1], RGB[2], 1.0f)); // not clamp out of gamut color.
    }
}
