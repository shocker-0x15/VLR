#include "hip/hip_runtime.h"
﻿#include "../shared/kernel_common.h"

namespace vlr {
    using namespace shared;

    CUDA_DEVICE_FUNCTION DirectionType sideTest(const Normal3D &ng, const Vector3D &d0, const Vector3D &d1) {
        bool reflect = dot(Vector3D(ng), d0) * dot(Vector3D(ng), d1) > 0;
        return DirectionType::AllFreq() | (reflect ? DirectionType::Reflection() : DirectionType::Transmission());
    }



    class FresnelConductor {
        SampledSpectrum m_eta;
        SampledSpectrum m_k;

    public:
        CUDA_DEVICE_FUNCTION FresnelConductor(const SampledSpectrum &eta, const SampledSpectrum &k) :
            m_eta(eta), m_k(k) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluate(float cosEnter) const {
            cosEnter = std::fabs(cosEnter);
            float cosEnter2 = cosEnter * cosEnter;
            SampledSpectrum _2EtaCosEnter = 2.0f * m_eta * cosEnter;
            SampledSpectrum tmp_f = m_eta * m_eta + m_k * m_k;
            SampledSpectrum tmp = tmp_f * cosEnter2;
            SampledSpectrum Rparl2 = (tmp - _2EtaCosEnter + 1) / (tmp + _2EtaCosEnter + 1);
            SampledSpectrum Rperp2 = (tmp_f - _2EtaCosEnter + cosEnter2) / (tmp_f + _2EtaCosEnter + cosEnter2);

            return (Rparl2 + Rperp2) / 2.0f;
        }
        CUDA_DEVICE_FUNCTION float evaluate(float cosEnter, uint32_t wlIdx) const {
            cosEnter = std::fabs(cosEnter);
            float cosEnter2 = cosEnter * cosEnter;
            float _2EtaCosEnter = 2.0f * m_eta[wlIdx] * cosEnter;
            float tmp_f = m_eta[wlIdx] * m_eta[wlIdx] + m_k[wlIdx] * m_k[wlIdx];
            float tmp = tmp_f * cosEnter2;
            float Rparl2 = (tmp - _2EtaCosEnter + 1) / (tmp + _2EtaCosEnter + 1);
            float Rperp2 = (tmp_f - _2EtaCosEnter + cosEnter2) / (tmp_f + _2EtaCosEnter + cosEnter2);

            return (Rparl2 + Rperp2) / 2.0f;
        }
    };



    class FresnelDielectric {
        SampledSpectrum m_etaExt;
        SampledSpectrum m_etaInt;

    public:
        CUDA_DEVICE_FUNCTION FresnelDielectric(const SampledSpectrum &etaExt, const SampledSpectrum &etaInt) : m_etaExt(etaExt), m_etaInt(etaInt) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum etaExt() const { return m_etaExt; }
        CUDA_DEVICE_FUNCTION SampledSpectrum etaInt() const { return m_etaInt; }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluate(float cosEnter) const {
            cosEnter = clamp(cosEnter, -1.0f, 1.0f);

            bool entering = cosEnter > 0.0f;
            const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;

            SampledSpectrum sinExit = eEnter / eExit * std::sqrt(std::fmax(0.0f, 1.0f - cosEnter * cosEnter));
            SampledSpectrum ret = SampledSpectrum::Zero();
            cosEnter = std::fabs(cosEnter);
            for (int i = 0; i < SampledSpectrum::NumComponents(); ++i) {
                if (sinExit[i] >= 1.0f) {
                    ret[i] = 1.0f;
                }
                else {
                    float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit[i] * sinExit[i]));
                    ret[i] = evalF(eEnter[i], eExit[i], cosEnter, cosExit);
                }
            }
            return ret;
        }
        CUDA_DEVICE_FUNCTION float evaluate(float cosEnter, uint32_t wlIdx) const {
            cosEnter = clamp(cosEnter, -1.0f, 1.0f);

            bool entering = cosEnter > 0.0f;
            const float &eEnter = entering ? m_etaExt[wlIdx] : m_etaInt[wlIdx];
            const float &eExit = entering ? m_etaInt[wlIdx] : m_etaExt[wlIdx];

            float sinExit = eEnter / eExit * std::sqrt(std::fmax(0.0f, 1.0f - cosEnter * cosEnter));
            cosEnter = std::fabs(cosEnter);
            if (sinExit >= 1.0f) {
                return 1.0f;
            }
            else {
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit * sinExit));
                return evalF(eEnter, eExit, cosEnter, cosExit);
            }
        }

        CUDA_DEVICE_FUNCTION static float evalF(float etaEnter, float etaExit, float cosEnter, float cosExit);
    };

    CUDA_DEVICE_FUNCTION float FresnelDielectric::evalF(float etaEnter, float etaExit, float cosEnter, float cosExit) {
        float Rparl = ((etaExit * cosEnter) - (etaEnter * cosExit)) / ((etaExit * cosEnter) + (etaEnter * cosExit));
        float Rperp = ((etaEnter * cosEnter) - (etaExit * cosExit)) / ((etaEnter * cosEnter) + (etaExit * cosExit));
        return (Rparl * Rparl + Rperp * Rperp) / 2.0f;
    }



    class FresnelSchlick {
        // assume vacuum-dielectric interface
        float m_F0;

    public:
        CUDA_DEVICE_FUNCTION FresnelSchlick(float F0) : m_F0(F0) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluate(float cosEnter) const {
            bool entering = cosEnter >= 0;
            float cosEval = cosEnter;
            if (!entering) {
                float sqrtF0 = std::sqrt(m_F0);
                float etaExit = (1 + sqrtF0) / (1 - sqrtF0);
                float invRelIOR = 1.0f / etaExit;
                float sinExit2 = invRelIOR * invRelIOR * std::fmax(0.0f, 1.0f - cosEnter * cosEnter);
                if (sinExit2 > 1.0f) {
                    return SampledSpectrum::One();
                }
                cosEval = std::sqrt(1 - sinExit2);
            }
            return SampledSpectrum(m_F0 + (1.0f - m_F0) * pow5(1 - cosEval));
        }
    };



    class GGXMicrofacetDistribution {
        float m_alpha_gx;
        float m_alpha_gy;
        float m_cosRt;
        float m_sinRt;

    public:
        CUDA_DEVICE_FUNCTION GGXMicrofacetDistribution(float alpha_gx, float alpha_gy, float rotation) :
            m_alpha_gx(alpha_gx), m_alpha_gy(alpha_gy) {
            ::vlr::sincos(rotation, &m_sinRt, &m_cosRt);
        }

        CUDA_DEVICE_FUNCTION float evaluate(const Normal3D &m) {
            Normal3D mr = Normal3D(m_cosRt * m.x + m_sinRt * m.y,
                                   -m_sinRt * m.x + m_cosRt * m.y,
                                   m.z);

            if (mr.z <= 0)
                return 0.0f;
            float temp = pow2(mr.x / m_alpha_gx) + pow2(mr.y / m_alpha_gy) + pow2(mr.z);
            return 1.0f / (VLR_M_PI * m_alpha_gx * m_alpha_gy * pow2(temp));
        }

        CUDA_DEVICE_FUNCTION float evaluateSmithG1(const Vector3D &v, const Normal3D &m) {
            Vector3D vr = Vector3D(m_cosRt * v.x + m_sinRt * v.y,
                                   -m_sinRt * v.x + m_cosRt * v.y,
                                   v.z);

            float alpha_g2_tanTheta2 = (pow2(vr.x * m_alpha_gx) + pow2(vr.y * m_alpha_gy)) / pow2(vr.z);
            float Lambda = (-1 + std::sqrt(1 + alpha_g2_tanTheta2)) / 2;
            float chi = (dot(v, m) / v.z) > 0 ? 1 : 0;
            return chi / (1 + Lambda);
        }

        CUDA_DEVICE_FUNCTION float evaluateHeightCorrelatedSmithG(const Vector3D &v1, const Vector3D &v2, const Normal3D &m) {
            Vector3D v1r = Vector3D(m_cosRt * v1.x + m_sinRt * v1.y,
                                    -m_sinRt * v1.x + m_cosRt * v1.y,
                                    v1.z);
            Vector3D v2r = Vector3D(m_cosRt * v2.x + m_sinRt * v2.y,
                                    -m_sinRt * v2.x + m_cosRt * v2.y,
                                    v2.z);

            float alpha_g2_tanTheta2_1 = (pow2(v1r.x * m_alpha_gx) + pow2(v1r.y * m_alpha_gy)) / pow2(v1r.z);
            float alpha_g2_tanTheta2_2 = (pow2(v2r.x * m_alpha_gx) + pow2(v2r.y * m_alpha_gy)) / pow2(v2r.z);
            float Lambda1 = (-1 + std::sqrt(1 + alpha_g2_tanTheta2_1)) / 2;
            float Lambda2 = (-1 + std::sqrt(1 + alpha_g2_tanTheta2_2)) / 2;
            float chi1 = (dot(v1, m) / v1.z) > 0 ? 1 : 0;
            float chi2 = (dot(v2, m) / v2.z) > 0 ? 1 : 0;
            return chi1 * chi2 / (1 + Lambda1 + Lambda2);
        }

        CUDA_DEVICE_FUNCTION float sample(const Vector3D &v, float u0, float u1, Normal3D* m, float* normalPDF) {
            Vector3D vr = Vector3D(m_cosRt * v.x + m_sinRt * v.y,
                                   -m_sinRt * v.x + m_cosRt * v.y,
                                   v.z);

            // stretch view
            Vector3D sv = normalize(Vector3D(m_alpha_gx * vr.x, m_alpha_gy * vr.y, vr.z));

            // orthonormal basis
            //        Vector3D T1 = (sv.z < 0.9999f) ? normalize(cross(sv, Vector3D::Ez)) : Vector3D::Ex;
            //        Vector3D T2 = cross(T1, sv);
            float distIn2D = std::sqrt(sv.x * sv.x + sv.y * sv.y);
            float recDistIn2D = 1.0f / distIn2D;
            Vector3D T1 = (sv.z < 0.9999f) ? Vector3D(sv.y * recDistIn2D, -sv.x * recDistIn2D, 0) : Vector3D::Ex();
            Vector3D T2 = Vector3D(T1.y * sv.z, -T1.x * sv.z, distIn2D);

            // sample point with polar coordinates (r, phi)
            float a = 1.0f / (1.0f + sv.z);
            float r = std::sqrt(u0);
            float phi = VLR_M_PI * ((u1 < a) ? u1 / a : 1 + (u1 - a) / (1.0f - a));
            float sinPhi, cosPhi;
            ::vlr::sincos(phi, &sinPhi, &cosPhi);
            float P1 = r * cosPhi;
            float P2 = r * sinPhi * ((u1 < a) ? 1.0f : sv.z);

            // compute normal
            Normal3D mr = P1 * T1 + P2 * T2 + std::sqrt(1.0f - P1 * P1 - P2 * P2) * sv;

            // unstretch
            mr = normalize(Normal3D(m_alpha_gx * mr.x, m_alpha_gy * mr.y, mr.z));

            float D = evaluate(mr);
            *normalPDF = evaluateSmithG1(vr, mr) * absDot(vr, mr) * D / std::fabs(vr.z);

            *m = Normal3D(m_cosRt * mr.x - m_sinRt * mr.y,
                          m_sinRt * mr.x + m_cosRt * mr.y,
                          mr.z);

            return D;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDF(const Vector3D &v, const Normal3D &m) {
            return evaluateSmithG1(v, m) * absDot(v, m) * evaluate(m) / std::fabs(v.z);
        }
    };



    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum BSDF_getBaseColor(
        const uint32_t* params) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.getBaseColor();
    }

    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION bool BSDF_matches(
        const uint32_t* params,
        DirectionType flags) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.matches(flags);
    }

    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum BSDF_sampleInternal(
        const uint32_t* params,
        const BSDFQuery &query, float uComponent, const float uDir[2],
        BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.sampleInternal(query, uComponent, uDir, result, revResult);
    }

    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum BSDF_evaluateInternal(
        const uint32_t* params,
        const BSDFQuery &query, const Vector3D &dirLocal, SampledSpectrum* revValue = nullptr) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.evaluateInternal(query, dirLocal, revValue);
    }

    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION float BSDF_evaluatePDFInternal(
        const uint32_t* params,
        const BSDFQuery &query, const Vector3D &dirLocal, float* revValue = nullptr) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.evaluatePDFInternal(query, dirLocal, revValue);
    }

    template <typename BSDFType>
    CUDA_DEVICE_FUNCTION float BSDF_weightInternal(
        const uint32_t* params,
        const BSDFQuery &query) {
        auto &p = *reinterpret_cast<const BSDFType*>(params);
        return p.weightInternal(query);
    }



#define DEFINE_BSDF_CALLABLE_PROGRAMS(BSDF)\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(BSDF ## _getBaseColor)(\
        const uint32_t* params) {\
        return BSDF_getBaseColor<BSDF>(params);\
    }\
    RT_CALLABLE_PROGRAM bool RT_DC_NAME(BSDF ## _matches)(\
        const uint32_t* params,\
        DirectionType flags) {\
        return BSDF_matches<BSDF>(params, flags);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(BSDF ## _sampleInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, float uComponent, const float uDir[2],\
        BSDFQueryResult* result) {\
        return BSDF_sampleInternal<BSDF>(params, query, uComponent, uDir, result);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(BSDF ## _sampleWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, float uComponent, const float uDir[2],\
        BSDFQueryResult* result, BSDFQueryReverseResult* revResult) {\
        return BSDF_sampleInternal<BSDF>(params, query, uComponent, uDir, result, revResult);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(BSDF ## _evaluateInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal) {\
        return BSDF_evaluateInternal<BSDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(BSDF ## _evaluateWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal, SampledSpectrum* revValue) {\
        return BSDF_evaluateInternal<BSDF>(params, query, dirLocal, revValue);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(BSDF ## _evaluatePDFInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal) {\
        return BSDF_evaluatePDFInternal<BSDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(BSDF ## _evaluatePDFWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal, float* revValue) {\
        return BSDF_evaluatePDFInternal<BSDF>(params, query, dirLocal, revValue);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(BSDF ## _weightInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query) {\
        return BSDF_weightInternal<BSDF>(params, query);\
    }




    template <typename EDFType>
    CUDA_DEVICE_FUNCTION bool EDF_matches(
        const uint32_t* params,
        DirectionType flags) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.matches(flags);
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_sampleInternal(
        const uint32_t* params,
        const EDFQuery &query, float uComponent, const float uDir[2],
        EDFQueryResult* result) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.sampleInternal(query, uComponent, uDir, result);
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_evaluateEmittanceInternal(
        const uint32_t* params) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.evaluateEmittanceInternal();
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_evaluateInternal(
        const uint32_t* params,
        const EDFQuery &query, const Vector3D &dirLocal) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.evaluateInternal(query, dirLocal);
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION float EDF_evaluatePDFInternal(
        const uint32_t* params,
        const EDFQuery &query, const Vector3D &dirLocal) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.evaluatePDFInternal(query, dirLocal);
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION float EDF_weightInternal(
        const uint32_t* params,
        const EDFQuery &query) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.weightInternal(query);
    }



#define DEFINE_EDF_CALLABLE_PROGRAMS(EDF)\
    RT_CALLABLE_PROGRAM bool RT_DC_NAME(EDF ## _matches)(\
        const uint32_t* params,\
        DirectionType flags) {\
        return EDF_matches<EDF>(params, flags);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _sampleInternal)(\
        const uint32_t* params,\
        const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) {\
        return EDF_sampleInternal<EDF>(params, query, uComponent, uDir, result);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _evaluateEmittanceInternal)(\
        const uint32_t* params) {\
        return EDF_evaluateEmittanceInternal<EDF>(params);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _evaluateInternal)(\
        const uint32_t* params, const EDFQuery &query, const Vector3D &dirLocal) {\
        return EDF_evaluateInternal<EDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(EDF ## _evaluatePDFInternal)(\
        const uint32_t* params,\
        const EDFQuery &query, const Vector3D &dirLocal) {\
        return EDF_evaluatePDFInternal<EDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(EDF ## _weightInternal)(\
        const uint32_t* params,\
        const EDFQuery &query) {\
        return EDF_weightInternal<EDF>(params, query);\
    }



    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_as_BSDF_getBaseColor(
        const uint32_t* params) {
        return SampledSpectrum::Zero();
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION bool EDF_as_BSDF_matches(
        const uint32_t* params,
        DirectionType flags) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        return p.matches(flags);
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_as_BSDF_sampleInternal(
        const uint32_t* params,
        const BSDFQuery &query, float uComponent, const float uDir[2],
        BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        EDFQuery edfQuery(query.dirTypeFilter, query.wlHint);
        EDFQueryResult edfResult;
        SampledSpectrum ret = p.sampleInternal(edfQuery, uComponent, uDir, &edfResult);
        result->dirLocal = edfResult.dirLocal;
        result->dirPDF = edfResult.dirPDF;
        result->sampledType = edfResult.sampledType;
        if (revResult) {
            revResult->value = SampledSpectrum::Zero();
            revResult->dirPDF = 0.0f;
        }
        return ret;
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION SampledSpectrum EDF_as_BSDF_evaluateInternal(
        const uint32_t* params,
        const BSDFQuery &query, const Vector3D &dirLocal, SampledSpectrum* revValue = nullptr) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        EDFQuery edfQuery(query.dirTypeFilter, query.wlHint);
        SampledSpectrum ret = p.evaluateInternal(edfQuery, dirLocal);
        if (revValue)
            *revValue = SampledSpectrum::Zero();
        return ret;
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION float EDF_as_BSDF_evaluatePDFInternal(
        const uint32_t* params,
        const BSDFQuery &query, const Vector3D &dirLocal, float* revValue = nullptr) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        EDFQuery edfQuery(query.dirTypeFilter, query.wlHint);
        float ret = p.evaluatePDFInternal(edfQuery, dirLocal);
        if (revValue)
            *revValue = 0.0f;
        return ret;
    }

    template <typename EDFType>
    CUDA_DEVICE_FUNCTION float EDF_as_BSDF_weightInternal(
        const uint32_t* params,
        const BSDFQuery &query) {
        auto &p = *reinterpret_cast<const EDFType*>(params);
        EDFQuery edfQuery(query.dirTypeFilter, query.wlHint);
        return p.weightInternal(edfQuery);
    }



#define DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(EDF)\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _as_BSDF_getBaseColor)(\
        const uint32_t* params) {\
        return EDF_as_BSDF_getBaseColor<EDF>(params);\
    }\
    RT_CALLABLE_PROGRAM bool RT_DC_NAME(EDF ## _as_BSDF_matches)(\
        const uint32_t* params,\
        DirectionType flags) {\
        return EDF_as_BSDF_matches<EDF>(params, flags);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _as_BSDF_sampleInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, float uComponent, const float uDir[2],\
        BSDFQueryResult* result) {\
        return EDF_as_BSDF_sampleInternal<EDF>(params, query, uComponent, uDir, result);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _as_BSDF_sampleWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, float uComponent, const float uDir[2],\
        BSDFQueryResult* result, BSDFQueryReverseResult* revResult) {\
        return EDF_as_BSDF_sampleInternal<EDF>(params, query, uComponent, uDir, result, revResult);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _as_BSDF_evaluateInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal) {\
        return EDF_as_BSDF_evaluateInternal<EDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM SampledSpectrum RT_DC_NAME(EDF ## _as_BSDF_evaluateWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal, SampledSpectrum* revValue) {\
        return EDF_as_BSDF_evaluateInternal<EDF>(params, query, dirLocal, revValue);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(EDF ## _as_BSDF_evaluatePDFInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal) {\
        return EDF_as_BSDF_evaluatePDFInternal<EDF>(params, query, dirLocal);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(EDF ## _as_BSDF_evaluatePDFWithRevInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query, const Vector3D &dirLocal, float* revValue) {\
        return EDF_as_BSDF_evaluatePDFInternal<EDF>(params, query, dirLocal, revValue);\
    }\
    RT_CALLABLE_PROGRAM float RT_DC_NAME(EDF ## _as_BSDF_weightInternal)(\
        const uint32_t* params,\
        const BSDFQuery &query) {\
        return EDF_as_BSDF_weightInternal<EDF>(params, query);\
    }



    class NullBSDF {
    public:
        CUDA_DEVICE_FUNCTION NullBSDF() {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            return false;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            result->dirPDF = 0.0f;
            if (revResult)
                revResult->dirPDF = 0.0f;
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            if (revValue)
                *revValue = SampledSpectrum::Zero();
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (*revValue)
                *revValue = 0.0f;
            return 0.0f;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            return 0.0f;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(NullBSDF_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        return 0;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(NullBSDF)



    class MatteBRDF {
        SampledSpectrum m_albedo;
        float m_roughness;

    public:
        CUDA_DEVICE_FUNCTION MatteBRDF(const SampledSpectrum &albedo, float roughness) :
            m_albedo(albedo), m_roughness(roughness) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_albedo;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType m_type = DirectionType::Reflection() | DirectionType::LowFreq();
            return m_type.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            result->dirLocal = cosineSampleHemisphere(uDir[0], uDir[1]);
            result->dirPDF = result->dirLocal.z / VLR_M_PI;
            result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();
            result->dirLocal.z *= query.dirLocal.z >= 0 ? 1 : -1;
            SampledSpectrum fsValue = m_albedo / VLR_M_PI;
            if (revResult) {
                revResult->value = fsValue;
                revResult->dirPDF = std::fabs(query.dirLocal.z) / VLR_M_PI;
            }

            return fsValue;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            if (query.dirLocal.z * dirLocal.z <= 0.0f) {
                SampledSpectrum fs = SampledSpectrum::Zero();
                if (revValue)
                    *revValue = fs;
                return fs;
            }
            SampledSpectrum fsValue = m_albedo / VLR_M_PI;
            if (revValue)
                *revValue = fsValue;

            return fsValue;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (query.dirLocal.z * dirLocal.z <= 0.0f) {
                if (revValue)
                    *revValue = 0.0f;
                return 0.0f;
            }
            float pdfValue = std::fabs(dirLocal.z) / VLR_M_PI;
            if (revValue)
                *revValue = std::fabs(query.dirLocal.z) / VLR_M_PI;

            return pdfValue;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            return m_albedo.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(MatteSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<MatteBRDF*>(params);
        auto &mat = *reinterpret_cast<const MatteSurfaceMaterial*>(matDesc);

        p = MatteBRDF(calcNode(mat.nodeAlbedo, mat.immAlbedo, surfPt, wls), 0.0f);

        return sizeof(MatteBRDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(MatteBRDF)



    class SpecularBRDF {
        SampledSpectrum m_coeffR;
        SampledSpectrum m_eta;
        SampledSpectrum m_k;

    public:
        CUDA_DEVICE_FUNCTION SpecularBRDF(
            const SampledSpectrum &coeffR, const SampledSpectrum &eta, const SampledSpectrum &k) :
        m_coeffR(coeffR), m_eta(eta), m_k(k) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_coeffR;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType;
            if constexpr (usePathSpaceRegularization)
                dirType = DirectionType::Reflection() | DirectionType::HighFreq();
            else
                dirType = DirectionType::Reflection() | DirectionType::Delta0D();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            float regFactor = 1.0f;
            DirectionType dirType = DirectionType::Delta0D();
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                regFactor = computeRegularizationFactor(&cosEpsilon);
                dirType = DirectionType::HighFreq();
            }

            FresnelConductor fresnel(m_eta, m_k);

            result->dirLocal = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
            result->dirPDF = regFactor;
            result->sampledType = DirectionType::Reflection() | dirType;
            SampledSpectrum ret = m_coeffR * fresnel.evaluate(query.dirLocal.z) *
                (regFactor / std::fabs(query.dirLocal.z));

            if (revResult) {
                revResult->value = ret;
                revResult->dirPDF = regFactor;
            }

            return ret;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                SampledSpectrum ret = SampledSpectrum::Zero();
                Vector3D dirReflected = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
                if (dot(dirLocal, dirReflected) >= cosEpsilon) {
                    FresnelConductor fresnel(m_eta, m_k);
                    ret = m_coeffR * fresnel.evaluate(query.dirLocal.z) *
                        (regFactor / std::fabs(query.dirLocal.z));
                }

                if (revValue)
                    *revValue = ret;

                return ret;
            }
            else {
                if (revValue)
                    *revValue = SampledSpectrum::Zero();
                return SampledSpectrum::Zero();
            }
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                float ret = 0.0f;
                Vector3D dirReflected = Vector3D(-query.dirLocal.x, -query.dirLocal.y, query.dirLocal.z);
                if (dot(dirLocal, dirReflected) >= cosEpsilon)
                    ret = regFactor;

                if (revValue)
                    *revValue = ret;

                return ret;
            }
            else {
                if (revValue)
                    *revValue = 0.0f;
                return 0.0f;
            }
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            FresnelDielectric fresnel(m_eta, m_k);
            float weight = (m_coeffR * fresnel.evaluate(query.dirLocal.z)).importance(query.wlHint);

            return weight;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(SpecularReflectionSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<SpecularBRDF*>(params);
        auto &mat = *reinterpret_cast<const SpecularReflectionSurfaceMaterial*>(matDesc);

        p = SpecularBRDF(
            calcNode(mat.nodeCoeffR, mat.immCoeffR, surfPt, wls),
            calcNode(mat.nodeEta, mat.immEta, surfPt, wls),
            calcNode(mat.node_k, mat.imm_k, surfPt, wls));

        return sizeof(SpecularBRDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(SpecularBRDF)



    class SpecularBSDF {
        SampledSpectrum m_coeff;
        SampledSpectrum m_etaExt;
        SampledSpectrum m_etaInt;
        bool m_dispersive;

    public:
        CUDA_DEVICE_FUNCTION SpecularBSDF(
            const SampledSpectrum &coeff, const SampledSpectrum &etaExt, const SampledSpectrum &etaInt,
            bool dispersive) :
            m_coeff(coeff), m_etaExt(etaExt), m_etaInt(etaInt), m_dispersive(dispersive) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_coeff;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType;
            if constexpr (usePathSpaceRegularization)
                dirType = DirectionType::WholeSphere() | DirectionType::HighFreq();
            else
                dirType = DirectionType::WholeSphere() | DirectionType::Delta0D();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
            FresnelDielectric fresnel(eEnter, eExit);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

            float regFactor = 1.0f;
            DirectionType dirType = DirectionType::Delta0D();
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                regFactor = computeRegularizationFactor(&cosEpsilon);
                dirType = DirectionType::HighFreq();
            }

            SampledSpectrum F = fresnel.evaluate(dirV.z);
            float reflectProb = F.importance(query.wlHint);
            if (query.dirTypeFilter.isReflection())
                reflectProb = 1.0f;
            if (query.dirTypeFilter.isTransmission())
                reflectProb = 0.0f;
            if (uComponent < reflectProb) {
                if (dirV.z == 0.0f) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                Vector3D dirL = Vector3D(-dirV.x, -dirV.y, dirV.z);
                result->dirLocal = entering ? dirL : -dirL;
                result->dirPDF = reflectProb * regFactor;
                result->sampledType = DirectionType::Reflection() | dirType;
                SampledSpectrum ret = m_coeff * F * (regFactor / std::fabs(dirV.z));

                if (revResult) {
                    revResult->value = ret;
                    revResult->dirPDF = result->dirPDF;
                }

                return ret;
            }
            else {
                float sinEnter2 = 1.0f - dirV.z * dirV.z;
                float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];// reciprocal of relative IOR.
                float sinExit2 = recRelIOR * recRelIOR * sinEnter2;

                if (sinExit2 >= 1.0f) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit2));
                Vector3D dirL = Vector3D(recRelIOR * -dirV.x, recRelIOR * -dirV.y, -cosExit);
                result->dirLocal = entering ? dirL : -dirL;
                result->dirPDF = (1.0f - reflectProb) * regFactor;
                result->sampledType =
                    DirectionType::Transmission() | dirType |
                    (m_dispersive ? DirectionType::Dispersive() : DirectionType());

                SampledSpectrum ret = SampledSpectrum::Zero();
                ret[query.wlHint] = m_coeff[query.wlHint] * (1.0f - F[query.wlHint]) * regFactor;

                if (revResult) {
                    float revSqueezeFactor = 1.0f;
                    if (static_cast<TransportMode>(query.transportMode) == TransportMode::Importance)
                        revSqueezeFactor = pow2(eExit[query.wlHint] / eEnter[query.wlHint]);
                    SampledSpectrum revRet = ret;
                    revRet[query.wlHint] *= revSqueezeFactor / std::fabs(query.dirLocal.z);
                    revResult->value = revRet;
                    revResult->dirPDF = revSqueezeFactor * result->dirPDF;
                }

                float squeezeFactor = 1.0f;
                if (static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance)
                    squeezeFactor *= pow2(eEnter[query.wlHint] / eExit[query.wlHint]);
                ret[query.wlHint] *= squeezeFactor / std::fabs(cosExit);
                result->dirPDF *= squeezeFactor;

                return ret;
            }
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            if constexpr (usePathSpaceRegularization) {
                bool entering = query.dirLocal.z >= 0.0f;

                const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
                const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
                FresnelDielectric fresnel(eEnter, eExit);

                Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                SampledSpectrum F = fresnel.evaluate(dirV.z);

                Vector3D dirReflected = Vector3D(-dirV.x, -dirV.y, dirV.z);
                if (!entering)
                    dirReflected = -dirReflected;
                if (dot(dirLocal, dirReflected) >= cosEpsilon) {
                    SampledSpectrum ret = m_coeff * F * (regFactor / std::fabs(dirV.z));
                    if (revValue)
                        *revValue = ret;
                    return ret;
                }

                float sinEnter2 = 1.0f - dirV.z * dirV.z;
                float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];// reciprocal of relative IOR.
                float sinExit2 = recRelIOR * recRelIOR * sinEnter2;

                SampledSpectrum ret = SampledSpectrum::Zero();
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit2));
                Vector3D dirRefracted = Vector3D(recRelIOR * -dirV.x, recRelIOR * -dirV.y, -cosExit);
                if (!entering)
                    dirRefracted = -dirRefracted;
                if (dot(dirLocal, dirRefracted) >= cosEpsilon && cosExit != 0.0f) {
                    ret[query.wlHint] = m_coeff[query.wlHint] * (1.0f - F[query.wlHint]) * regFactor;

                    if (revValue) {
                        float revSqueezeFactor = 1.0f;
                        if (static_cast<TransportMode>(query.transportMode) == TransportMode::Importance)
                            revSqueezeFactor = pow2(eExit[query.wlHint] / eEnter[query.wlHint]);
                        SampledSpectrum revRet = ret;
                        revRet[query.wlHint] *= revSqueezeFactor / std::fabs(query.dirLocal.z);
                        *revValue = revRet;
                    }

                    float squeezeFactor = 1.0f;
                    if (static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance)
                        squeezeFactor *= pow2(eEnter[query.wlHint] / eExit[query.wlHint]);
                    ret[query.wlHint] *= squeezeFactor / std::fabs(cosExit);
                }
                else {
                    if (revValue)
                        *revValue = SampledSpectrum::Zero();
                }

                return ret;
            }
            else {
                if (revValue)
                    *revValue = SampledSpectrum::Zero();
                return SampledSpectrum::Zero();
            }
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            if constexpr (usePathSpaceRegularization) {
                bool entering = query.dirLocal.z >= 0.0f;

                const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
                const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
                FresnelDielectric fresnel(eEnter, eExit);

                Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                SampledSpectrum F = fresnel.evaluate(dirV.z);
                float reflectProb = F.importance(query.wlHint);
                if (query.dirTypeFilter.isReflection())
                    reflectProb = 1.0f;
                if (query.dirTypeFilter.isTransmission())
                    reflectProb = 0.0f;

                if (dirV.z == 0.0f)
                    return 0.0f;
                Vector3D dirReflected = Vector3D(-dirV.x, -dirV.y, dirV.z);
                if (!entering)
                    dirReflected = -dirReflected;
                if (dot(dirLocal, dirReflected) >= cosEpsilon) {
                    float ret = reflectProb * regFactor;
                    if (revValue)
                        *revValue = ret;
                    return ret;
                }

                float sinEnter2 = 1.0f - dirV.z * dirV.z;
                float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];// reciprocal of relative IOR.
                float sinExit2 = recRelIOR * recRelIOR * sinEnter2;

                if (sinExit2 >= 1.0f)
                    return 0.0f;
                float ret = 0.0f;
                float cosExit = std::sqrt(std::fmax(0.0f, 1.0f - sinExit2));
                Vector3D dirRefracted = Vector3D(recRelIOR * -dirV.x, recRelIOR * -dirV.y, -cosExit);
                if (!entering)
                    dirRefracted = -dirRefracted;
                if (dot(dirLocal, dirRefracted) >= cosEpsilon)
                    ret = (1.0f - reflectProb) * regFactor;

                if (revValue) {
                    float revSqueezeFactor = 1.0f;
                    if (static_cast<TransportMode>(query.transportMode) == TransportMode::Importance)
                        revSqueezeFactor = pow2(eExit[query.wlHint] / eEnter[query.wlHint]);
                    *revValue = revSqueezeFactor * ret;
                }

                float squeezeFactor = 1.0f;
                if (static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance)
                    squeezeFactor = pow2(eEnter[query.wlHint] / eExit[query.wlHint]);
                ret *= squeezeFactor;

                return ret;
            }
            else {
                return 0.0f;
            }
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            return m_coeff.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(SpecularScatteringSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<SpecularBSDF*>(params);
        auto &mat = *reinterpret_cast<const SpecularScatteringSurfaceMaterial*>(matDesc);

        p = SpecularBSDF(
            calcNode(mat.nodeCoeff, mat.immCoeff, surfPt, wls),
            calcNode(mat.nodeEtaExt, mat.immEtaExt, surfPt, wls),
            calcNode(mat.nodeEtaInt, mat.immEtaInt, surfPt, wls),
            !wls.singleIsSelected());

        return sizeof(SpecularBSDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(SpecularBSDF)



    class MicrofacetBRDF {
        SampledSpectrum m_eta;
        SampledSpectrum m_k;
        float m_alphaX;
        float m_alphaY;
        float m_rotation;

    public:
        CUDA_DEVICE_FUNCTION MicrofacetBRDF(
            const SampledSpectrum &eta, const SampledSpectrum &k,
            float alphaX, float alphaY, float rotation) :
        m_eta(eta), m_k(k), m_alphaX(alphaX), m_alphaY(alphaY), m_rotation(rotation) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            FresnelConductor fresnel(m_eta, m_k);

            return fresnel.evaluate(1.0f);
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType m_type = DirectionType::Reflection() | DirectionType::HighFreq();
            return m_type.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            FresnelConductor fresnel(m_eta, m_k);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

            // JP: ハーフベクトルをサンプルして、最終的な方向サンプルを生成する。
            // EN: sample a half vector, then generate a resulting direction sample based on it.
            Normal3D m;
            float mPDF;
            float D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
            float dotHV = dot(dirV, m);
            if (dotHV <= 0) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            Vector3D dirL = 2 * dotHV * m - dirV;
            result->dirLocal = entering ? dirL : -dirL;
            if (dirL.z * dirV.z <= 0) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            float commonPDFTerm = 1.0f / (4 * dotHV);
            result->dirPDF = commonPDFTerm * mPDF;
            result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

            SampledSpectrum F = fresnel.evaluate(dotHV);
            float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
            SampledSpectrum fs = F * D * G / (4 * dirV.z * dirL.z);

            if (revResult) {
                revResult->value = fs;
                revResult->dirPDF = commonPDFTerm * ggx.evaluatePDF(dirL, m);
            }

            //VLRAssert(fs.allFinite(), "fs: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, rDir: %s",
            //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return fs;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            bool entering = query.dirLocal.z >= 0.0f;

            FresnelConductor fresnel(m_eta, m_k);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;
            float dotNVdotNL = dirL.z * dirV.z;

            if (dotNVdotNL <= 0)
                return SampledSpectrum::Zero();

            Normal3D m = halfVector(dirV, dirL);
            float dotHV = dot(dirV, m);
            float D = ggx.evaluate(m);

            SampledSpectrum F = fresnel.evaluate(dotHV);
            float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
            SampledSpectrum fs = F * D * G / (4 * dotNVdotNL);

            if (revValue)
                *revValue = fs;

            //VLRAssert(fs.allFinite(), "fs: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
            //          fs.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return fs;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            FresnelConductor fresnel(m_eta, m_k);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;
            float dotNVdotNL = dirL.z * dirV.z;

            if (dotNVdotNL <= 0.0f)
                return 0.0f;

            Normal3D m = halfVector(dirV, dirL);
            float dotHV = dot(dirV, m);
            if (dotHV <= 0)
                return 0.0f;

            float mPDF = ggx.evaluatePDF(dirV, m);
            float commonPDFTerm = 1.0f / (4 * dotHV);
            float ret = commonPDFTerm * mPDF;

            if (revValue)
                *revValue = commonPDFTerm * ggx.evaluatePDF(dirL, m);

            //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
            //          "commonPDFTerm: %g, mPDF: %g, wlIdx: %u, qDir: %s, dir: %s",
            //          commonPDFTerm, mPDF, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

            return ret;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(
            const BSDFQuery &query) const {
            FresnelConductor fresnel(m_eta, m_k);

            float expectedDotHV = query.dirLocal.z;

            return fresnel.evaluate(expectedDotHV).importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(MicrofacetReflectionSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<MicrofacetBRDF*>(params);
        auto &mat = *reinterpret_cast<const MicrofacetReflectionSurfaceMaterial*>(matDesc);

        SampledSpectrum eta = calcNode(mat.nodeEta, mat.immEta, surfPt, wls);
        SampledSpectrum k = calcNode(mat.node_k, mat.imm_k, surfPt, wls);
        float3 roughnessAnisotropyRotation = 
            calcNode(mat.nodeRoughnessAnisotropyRotation,
                     make_float3(mat.immRoughness, mat.immAnisotropy, mat.immRotation),
                     surfPt, wls);
        float alpha = pow2(roughnessAnisotropyRotation.x);
        float aspect = std::sqrt(1.0f - 0.9f * roughnessAnisotropyRotation.y);
        float alphaX = std::fmax(0.001f, alpha / aspect);
        float alphaY = std::fmax(0.001f, alpha * aspect);
        float rotation = 2 * VLR_M_PI * roughnessAnisotropyRotation.z;

        p = MicrofacetBRDF(eta, k, alphaX, alphaY, rotation);

        return sizeof(MicrofacetBRDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(MicrofacetBRDF)



    class MicrofacetBSDF {
        SampledSpectrum m_coeff;
        SampledSpectrum m_etaExt;
        SampledSpectrum m_etaInt;
        float m_alphaX;
        float m_alphaY;
        float m_rotation;

    public:
        CUDA_DEVICE_FUNCTION MicrofacetBSDF(
            const SampledSpectrum &coeff, const SampledSpectrum &etaExt, const SampledSpectrum &etaInt,
            float alphaX, float alphaY, float rotation) :
        m_coeff(coeff), m_etaExt(etaExt), m_etaInt(etaInt),
        m_alphaX(alphaX), m_alphaY(alphaY), m_rotation(rotation) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_coeff;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType m_type = DirectionType::WholeSphere() | DirectionType::HighFreq();
            return m_type.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
            FresnelDielectric fresnel(eEnter, eExit);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

            // JP: ハーフベクトルをサンプルする。
            // EN: sample a half vector.
            Normal3D m;
            float mPDF;
            float D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
            float dotHV = dot(dirV, m);
            if (dotHV <= 0 || ::vlr::isnan(D)) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            // JP: サンプルしたハーフベクトルからフレネル項の値を計算して、反射か透過を選択する。
            // EN: calculate the Fresnel term using the sampled half vector, then select reflection or transmission.
            SampledSpectrum F = fresnel.evaluate(dotHV);
            float reflectProb = F.importance(query.wlHint);
            if (query.dirTypeFilter.isReflection())
                reflectProb = 1.0f;
            if (query.dirTypeFilter.isTransmission())
                reflectProb = 0.0f;
            if (uComponent < reflectProb) {
                // JP: 最終的な方向サンプルを生成する。
                // EN: calculate a resulting direction.
                Vector3D dirL = 2 * dotHV * m - dirV;
                result->dirLocal = entering ? dirL : -dirL;
                if (dirL.z * dirV.z <= 0) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                float commonPDFTerm = reflectProb / (4 * dotHV);
                result->dirPDF = commonPDFTerm * mPDF;
                result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

                float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
                SampledSpectrum ret = m_coeff * F * D * G / (4 * dirV.z * dirL.z);

                if (revResult) {
                    revResult->value = ret;
                    revResult->dirPDF = commonPDFTerm * ggx.evaluatePDF(dirL, m);
                }

                //VLRAssert(ret.allFinite(), "ret: %s, F: %g, %g, %g, G, %g, D: %g, wlIdx: %u, qDir: (%g, %g, %g), rDir: (%g, %g, %g)",
                //          ret.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, 
                //          dirV.x, dirV.y, dirV.z, dirL.x, dirL.y, dirL.z);

                return ret;
            }
            else {
                // JP: 最終的な方向サンプルを生成する。
                // EN: calculate a resulting direction.
                float recRelIOR = eEnter[query.wlHint] / eExit[query.wlHint];
                float innerRoot = 1 + recRelIOR * recRelIOR * (dotHV * dotHV - 1);
                if (innerRoot < 0) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                Vector3D dirL = (recRelIOR * dotHV - std::sqrt(innerRoot)) * m - recRelIOR * dirV;
                result->dirLocal = entering ? dirL : -dirL;
                if (dirL.z * dirV.z >= 0) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                float dotHL = dot(dirL, m);
                float commonPDFTerm = (1 - reflectProb) / pow2(eEnter[query.wlHint] * dotHV + eExit[query.wlHint] * dotHL);
                result->dirPDF = commonPDFTerm * mPDF * pow2(eExit[query.wlHint]) * std::fabs(dotHL);
                result->sampledType = DirectionType::Transmission() | DirectionType::HighFreq();

                // JP: マイクロファセットBSDFの各項の値を波長成分ごとに計算する。
                // EN: calculate the value of each term of the microfacet BSDF for each wavelength component.
                SampledSpectrum ret = SampledSpectrum::Zero();
                for (int wlIdx = 0; wlIdx < SampledSpectrum::NumComponents(); ++wlIdx) {
                    Normal3D m_wl = normalize(-(eEnter[wlIdx] * dirV + eExit[wlIdx] * dirL) * (entering ? 1 : -1));
                    float dotHV_wl = dot(dirV, m_wl);
                    float dotHL_wl = dot(dirL, m_wl);
                    float F_wl = fresnel.evaluate(dotHV_wl, wlIdx);
                    float G_wl = ggx.evaluateSmithG1(dirV, m_wl) * ggx.evaluateSmithG1(dirL, m_wl);
                    float D_wl = ggx.evaluate(m_wl);
                    ret[wlIdx] = std::fabs(dotHV_wl * dotHL_wl) * (1 - F_wl) * G_wl * D_wl / pow2(eEnter[wlIdx] * dotHV_wl + eExit[wlIdx] * dotHL_wl);

                    //VLRAssert(std::isfinite(ret[wlIdx]), "fs: %g, F: %g, G, %g, D: %g, wlIdx: %u, qDir: %s",
                    //          ret[wlIdx], F_wl, G_wl, D_wl, query.wlHint, dirV.toString().c_str());
                }
                ret /= std::fabs(dirV.z * dirL.z);
                ret *= m_coeff;
                ret *= static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance ?
                    pow2(eEnter) : pow2(eExit); // adjoint: need to cancel eEnter^2 / eExit^2 => eEnter^2 * (eExit^2 / eEnter^2)

                if (revResult) {
                    SampledSpectrum revRet = ret;
                    revResult->value = revRet;
                    revResult->dirPDF = commonPDFTerm * ggx.evaluatePDF(dirL, m) * pow2(eEnter[query.wlHint]) * std::fabs(dotHV);
                }

                //VLRAssert(ret.allFinite(), "fs: %s, wlIdx: %u, qDir: %s, rDir: %s",
                //          ret.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

                return ret;
            }
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            bool entering = query.dirLocal.z >= 0.0f;

            const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
            FresnelDielectric fresnel(eEnter, eExit);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;
            float dotNVdotNL = dirL.z * dirV.z;

            if (dotNVdotNL > 0 && query.dirTypeFilter.matches(DirectionType::Reflection() | DirectionType::AllFreq())) {
                Normal3D m = halfVector(dirV, dirL);
                float dotHV = dot(dirV, m);
                float D = ggx.evaluate(m);

                SampledSpectrum F = fresnel.evaluate(dotHV);
                float G = ggx.evaluateSmithG1(dirV, m) * ggx.evaluateSmithG1(dirL, m);
                SampledSpectrum ret = m_coeff * F * D * G / (4 * dotNVdotNL);

                if (revValue)
                    *revValue = ret;

                //VLRAssert(ret.allFinite(), "ret: %s, F: %s, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
                //          ret.toString().c_str(), F.toString().c_str(), G, D, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

                return ret;
            }
            else if (dotNVdotNL < 0 && query.dirTypeFilter.matches(DirectionType::Transmission() | DirectionType::AllFreq())) {
                SampledSpectrum ret = SampledSpectrum::Zero();
                for (int wlIdx = 0; wlIdx < SampledSpectrum::NumComponents(); ++wlIdx) {
                    Normal3D m_wl = normalize(-(eEnter[wlIdx] * dirV + eExit[wlIdx] * dirL) * (entering ? 1 : -1));
                    float dotHV_wl = dot(dirV, m_wl);
                    float dotHL_wl = dot(dirL, m_wl);
                    float F_wl = fresnel.evaluate(dotHV_wl, wlIdx);
                    float G_wl = ggx.evaluateSmithG1(dirV, m_wl) * ggx.evaluateSmithG1(dirL, m_wl);
                    float D_wl = ggx.evaluate(m_wl);
                    ret[wlIdx] = std::fabs(dotHV_wl * dotHL_wl) * (1 - F_wl) * G_wl * D_wl / pow2(eEnter[wlIdx] * dotHV_wl + eExit[wlIdx] * dotHL_wl);

                    //VLRAssert(std::isfinite(ret[wlIdx]), "fs: %g, F: %g, G, %g, D: %g, wlIdx: %u, qDir: %s, dir: %s",
                    //          ret[wlIdx], F_wl, G_wl, D_wl, query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());
                }
                ret /= std::fabs(dotNVdotNL);
                ret *= m_coeff;
                ret *= static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance ?
                    pow2(eEnter) : pow2(eExit); // !adjoint: eExit^2 * (eEnter / eExit)^2

                if (revValue)
                    *revValue = ret;

                //VLRAssert(ret.allFinite(), "fs: %s, wlIdx: %u, qDir: %s, dir: %s",
                //          ret.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

                return ret;
            }

            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            const SampledSpectrum &eEnter = entering ? m_etaExt : m_etaInt;
            const SampledSpectrum &eExit = entering ? m_etaInt : m_etaExt;
            FresnelDielectric fresnel(eEnter, eExit);

            GGXMicrofacetDistribution ggx(m_alphaX, m_alphaY, m_rotation);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;
            float dotNVdotNL = dirL.z * dirV.z;
            if (dotNVdotNL == 0)
                return 0.0f;

            Normal3D m;
            if (dotNVdotNL > 0)
                m = halfVector(dirV, dirL);
            else
                m = normalize(-(eEnter[query.wlHint] * dirV + eExit[query.wlHint] * dirL));
            float dotHV = dot(dirV, m);
            if (dotHV <= 0)
                return 0.0f;
            float mPDF = ggx.evaluatePDF(dirV, m);

            SampledSpectrum F = fresnel.evaluate(dotHV);
            float reflectProb = F.importance(query.wlHint);
            if (query.dirTypeFilter.isReflection())
                reflectProb = 1.0f;
            if (query.dirTypeFilter.isTransmission())
                reflectProb = 0.0f;
            if (dotNVdotNL > 0) {
                float commonPDFTerm = reflectProb / (4 * dotHV);

                if (revValue)
                    *revValue = commonPDFTerm * ggx.evaluatePDF(dirL, m);

                //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
                //          "commonPDFTerm: %g, mPDF: %g, F: %s, wlIdx: %u, qDir: %s, dir: %s",
                //          commonPDFTerm, mPDF, F.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

                return commonPDFTerm * mPDF;
            }
            else {
                float dotHL = dot(dirL, m);
                float commonPDFTerm = (1 - reflectProb) / pow2(eEnter[query.wlHint] * dotHV + eExit[query.wlHint] * dotHL);

                if (revValue)
                    *revValue = commonPDFTerm * ggx.evaluatePDF(dirL, m) * pow2(eEnter[query.wlHint]) * std::fabs(dotHV);

                //VLRAssert(std::isfinite(commonPDFTerm) && std::isfinite(mPDF),
                //          "commonPDFTerm: %g, mPDF: %g, F: %s, wlIdx: %u, qDir: %s, dir: %s",
                //          commonPDFTerm, mPDF, F.toString().c_str(), query.wlHint, dirV.toString().c_str(), dirL.toString().c_str());

                return commonPDFTerm * mPDF * pow2(eExit[query.wlHint]) * std::fabs(dotHL);
            }
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            return m_coeff.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(MicrofacetScatteringSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<MicrofacetBSDF*>(params);
        auto &mat = *reinterpret_cast<const MicrofacetScatteringSurfaceMaterial*>(matDesc);

        SampledSpectrum coeff = calcNode(mat.nodeCoeff, mat.immCoeff, surfPt, wls);
        SampledSpectrum etaExt = calcNode(mat.nodeEtaExt, mat.immEtaExt, surfPt, wls);
        SampledSpectrum etaInt = calcNode(mat.nodeEtaInt, mat.immEtaInt, surfPt, wls);
        float3 roughnessAnisotropyRotation = calcNode(mat.nodeRoughnessAnisotropyRotation,
                                                      make_float3(mat.immRoughness, mat.immAnisotropy, mat.immRotation),
                                                      surfPt, wls);
        float alpha = pow2(roughnessAnisotropyRotation.x);
        float aspect = std::sqrt(1 - 0.9f * roughnessAnisotropyRotation.y);
        float alphaX = std::fmax(0.001f, alpha / aspect);
        float alphaY = std::fmax(0.001f, alpha * aspect);
        float rotation = 2 * VLR_M_PI * roughnessAnisotropyRotation.z;

        p = MicrofacetBSDF(coeff, etaExt, etaInt, alphaX, alphaY, rotation);

        return sizeof(MicrofacetBSDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(MicrofacetBSDF)



    class LambertianBSDF {
        SampledSpectrum m_coeff;
        float m_F0;

    public:
        CUDA_DEVICE_FUNCTION LambertianBSDF(
            const SampledSpectrum &coeff, float F0) :
        m_coeff(coeff), m_F0(F0) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_coeff;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType m_type = DirectionType::WholeSphere() | DirectionType::LowFreq();
            return m_type.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            FresnelSchlick fresnel(m_F0);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = cosineSampleHemisphere(uDir[0], uDir[1]);
            result->dirPDF = dirL.z / VLR_M_PI;

            SampledSpectrum F = fresnel.evaluate(query.dirLocal.z);
            float reflectProb = F.importance(query.wlHint);
            if (query.dirTypeFilter.isReflection())
                reflectProb = 1.0f;
            if (query.dirTypeFilter.isTransmission())
                reflectProb = 0.0f;

            if (uComponent < reflectProb) {
                result->dirLocal = entering ? dirL : -dirL;
                result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();
                SampledSpectrum ret = F * m_coeff / VLR_M_PI;
                result->dirPDF *= reflectProb;

                if (revResult) {
                    revResult->value = ret;
                    revResult->dirPDF = reflectProb * dirV.z / VLR_M_PI;
                }

                return ret;
            }
            else {
                result->dirLocal = entering ? -dirL : dirL;
                result->sampledType = DirectionType::Transmission() | DirectionType::LowFreq();
                SampledSpectrum ret = (SampledSpectrum::One() - F) * m_coeff / VLR_M_PI;
                result->dirPDF *= (1 - reflectProb);

                if (revResult) {
                    revResult->value = ret;
                    revResult->dirPDF = (1 - reflectProb) * dirV.z / VLR_M_PI;
                }

                return ret;
            }
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            bool entering = query.dirLocal.z >= 0.0f;

            FresnelSchlick fresnel(m_F0);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;

            SampledSpectrum F = fresnel.evaluate(query.dirLocal.z);

            if (dirV.z * dirL.z > 0.0f) {
                SampledSpectrum ret = F * m_coeff / VLR_M_PI;
                if (revValue)
                    *revValue = ret;
                return ret;
            }
            else {
                SampledSpectrum ret = (SampledSpectrum::One() - F) * m_coeff / VLR_M_PI;
                if (revValue)
                    *revValue = ret;
                return ret;
            }
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            bool entering = query.dirLocal.z >= 0.0f;

            FresnelSchlick fresnel(m_F0);

            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;

            SampledSpectrum F = fresnel.evaluate(query.dirLocal.z);
            float reflectProb = F.importance(query.wlHint);
            if (query.dirTypeFilter.isReflection())
                reflectProb = 1.0f;
            if (query.dirTypeFilter.isTransmission())
                reflectProb = 0.0f;

            if (dirV.z * dirL.z > 0.0f) {
                float dirPDF = reflectProb * dirL.z / VLR_M_PI;
                if (revValue)
                    *revValue = reflectProb * dirV.z / VLR_M_PI;
                return dirPDF;
            }
            else {
                float dirPDF = (1 - reflectProb) * std::fabs(dirL.z) / VLR_M_PI;
                if (revValue)
                    *revValue = (1 - reflectProb) * dirV.z / VLR_M_PI;
                return dirPDF;
            }
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            return m_coeff.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(LambertianScatteringSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<LambertianBSDF*>(params);
        auto &mat = *reinterpret_cast<const LambertianScatteringSurfaceMaterial*>(matDesc);

        p = LambertianBSDF(
            calcNode(mat.nodeCoeff, mat.immCoeff, surfPt, wls),
            calcNode(mat.nodeF0, mat.immF0, surfPt, wls));

        return sizeof(LambertianBSDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(LambertianBSDF)



#define USE_HEIGHT_CORRELATED_SMITH

    class DiffuseAndSpecularBRDF {
        SampledSpectrum m_diffuseColor;
        SampledSpectrum m_specularF0Color;
        float m_roughness;

    public:
        CUDA_DEVICE_FUNCTION DiffuseAndSpecularBRDF(
            const SampledSpectrum &diffuseColor, const SampledSpectrum &specularF0Color, float roughness) :
            m_diffuseColor(diffuseColor), m_specularF0Color(specularF0Color), m_roughness(roughness) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            return m_diffuseColor + m_specularF0Color;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType m_type = DirectionType::Reflection() | DirectionType::LowFreq() | DirectionType::HighFreq();
            return m_type.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            float alpha = pow2(m_roughness);
            GGXMicrofacetDistribution ggx(alpha, alpha, 0.0f);

            bool entering = query.dirLocal.z >= 0.0f;
            Vector3D dirL;
            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

            float expectedF_D90 = 0.5f * m_roughness + 2 * m_roughness * pow2(query.dirLocal.z);
            float oneMinusDotVN5 = pow5(1 - dirV.z);
            float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
            float iBaseColor = m_diffuseColor.importance(query.wlHint) * pow2(expectedDiffuseFresnel) * lerp(1.0f, 1.0f / 1.51f, m_roughness);

            float expectedOneMinusDotVH5 = pow5(1 - dirV.z);
            float iSpecularF0 = m_specularF0Color.importance(query.wlHint);

            float diffuseWeight = iBaseColor;
            float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

            float weights[] = { diffuseWeight, specularWeight };
            float probSelection;
            float sumWeights = 0.0f;
            uint32_t component = sampleDiscrete(weights, 2, uComponent, &probSelection, &sumWeights, &uComponent);

            float diffuseDirPDF, specularDirPDF;
            SampledSpectrum fs;
            Normal3D m;
            float dotLH;
            float D;
            if (component == 0) {
                result->sampledType = DirectionType::Reflection() | DirectionType::LowFreq();

                // JP: コサイン分布からサンプルする。
                // EN: sample based on cosine distribution.
                dirL = cosineSampleHemisphere(uDir[0], uDir[1]);
                diffuseDirPDF = dirL.z / VLR_M_PI;

                // JP: 同じ方向サンプルを別の要素からサンプルする確率密度を求める。
                // EN: calculate PDFs to generate the sampled direction from the other distributions.
                m = halfVector(dirL, dirV);
                dotLH = dot(dirL, m);
                float commonPDFTerm = 1.0f / (4 * dotLH);
                specularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirV, m);

                D = ggx.evaluate(m);
            }
            else if (component == 1) {
                result->sampledType = DirectionType::Reflection() | DirectionType::HighFreq();

                // ----------------------------------------------------------------
                // JP: ベーススペキュラー層のマイクロファセット分布からサンプルする。
                // EN: sample based on the base specular microfacet distribution.
                float mPDF;
                D = ggx.sample(dirV, uDir[0], uDir[1], &m, &mPDF);
                float dotVH = dot(dirV, m);
                dotLH = dotVH;
                dirL = 2 * dotVH * m - dirV;
                if (dirL.z * dirV.z <= 0) {
                    result->dirPDF = 0.0f;
                    return SampledSpectrum::Zero();
                }
                float commonPDFTerm = 1.0f / (4 * dotLH);
                specularDirPDF = commonPDFTerm * mPDF;
                // ----------------------------------------------------------------

                // JP: 同じ方向サンプルを別の要素からサンプルする確率密度を求める。
                // EN: calculate PDFs to generate the sampled direction from the other distributions.
                diffuseDirPDF = dirL.z / VLR_M_PI;
            }

            float oneMinusDotLH5 = pow5(1 - dotLH);

#if defined(USE_HEIGHT_CORRELATED_SMITH)
            float G = ggx.evaluateHeightCorrelatedSmithG(dirL, dirV, m);
#else
            float G = ggx.evaluateSmithG1(dirL, m) * ggx.evaluateSmithG1(dirV, m);
#endif
            SampledSpectrum F = lerp(m_specularF0Color, SampledSpectrum::One(), oneMinusDotLH5);

            float microfacetDenom = 4 * dirL.z * dirV.z;
            SampledSpectrum specularValue = F * ((D * G) / microfacetDenom);

            float F_D90 = 0.5f * m_roughness + 2 * m_roughness * dotLH * dotLH;
            float oneMinusDotLN5 = pow5(1 - dirL.z);
            float diffuseFresnelOut = lerp(1.0f, F_D90, oneMinusDotVN5);
            float diffuseFresnelIn = lerp(1.0f, F_D90, oneMinusDotLN5);
            SampledSpectrum diffuseValue = m_diffuseColor * (diffuseFresnelOut * diffuseFresnelIn * lerp(1.0f, 1.0f / 1.51f, m_roughness) / VLR_M_PI);

            SampledSpectrum ret = diffuseValue + specularValue;

            result->dirLocal = entering ? dirL : -dirL;

            // PDF based on the single-sample model MIS.
            result->dirPDF = (diffuseDirPDF * diffuseWeight + specularDirPDF * specularWeight) / sumWeights;

            if (revResult) {
                float revDiffuseDirPDF = dirV.z / VLR_M_PI;
                float commonPDFTerm = 1.0f / (4 * dotLH);
                float revSpecularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirL, m);

                float revExpectedF_D90 = 0.5f * m_roughness + 2 * m_roughness * pow2(dirL.z);
                float revOneMinusDotVN5 = pow5(1 - dirL.z);
                float revExpectedDiffuseFresnel = lerp(1.0f, revExpectedF_D90, revOneMinusDotVN5);
                float revIBaseColor = m_diffuseColor.importance(query.wlHint) * pow2(revExpectedDiffuseFresnel) * lerp(1.0f, 1.0f / 1.51f, m_roughness);

                float revExpectedOneMinusDotVH5 = pow5(1 - dirL.z);
                float revISpecularF0 = m_specularF0Color.importance(query.wlHint);

                float revDiffuseWeight = revIBaseColor;
                float revSpecularWeight = lerp(revISpecularF0, 1.0f, revExpectedOneMinusDotVH5);

                revResult->value = ret;
                revResult->dirPDF =
                    (revDiffuseDirPDF * revDiffuseWeight + revSpecularDirPDF * revSpecularWeight) /
                    (revDiffuseWeight + revSpecularWeight);
            }

            return ret;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            float alpha = pow2(m_roughness);
            GGXMicrofacetDistribution ggx(alpha, alpha, 0.0f);

            if (dirLocal.z * query.dirLocal.z <= 0)
                return SampledSpectrum::Zero();

            bool entering = query.dirLocal.z >= 0.0f;
            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;

            Normal3D m = halfVector(dirL, dirV);
            float dotLH = dot(dirL, m);

            float oneMinusDotLH5 = pow5(1 - dotLH);

            float D = ggx.evaluate(m);
#if defined(USE_HEIGHT_CORRELATED_SMITH)
            float G = ggx.evaluateHeightCorrelatedSmithG(dirL, dirV, m);
#else
            float G = ggx.evaluateSmithG1(dirL, m) * ggx.evaluateSmithG1(dirV, m);
#endif
            SampledSpectrum F = lerp(m_specularF0Color, SampledSpectrum::One(), oneMinusDotLH5);

            float microfacetDenom = 4 * dirL.z * dirV.z;
            SampledSpectrum specularValue = F * ((D * G) / microfacetDenom);

            float F_D90 = 0.5f * m_roughness + 2 * m_roughness * pow2(dotLH);
            float oneMinusDotVN5 = pow5(1 - dirV.z);
            float oneMinusDotLN5 = pow5(1 - dirL.z);
            float diffuseFresnelOut = lerp(1.0f, F_D90, oneMinusDotVN5);
            float diffuseFresnelIn = lerp(1.0f, F_D90, oneMinusDotLN5);

            SampledSpectrum diffuseValue = m_diffuseColor * (diffuseFresnelOut * diffuseFresnelIn * lerp(1.0f, 1.0f / 1.51f, m_roughness) / VLR_M_PI);

            SampledSpectrum ret = diffuseValue + specularValue;

            if (revValue)
                *revValue = ret;

            return ret;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            float alpha = pow2(m_roughness);
            GGXMicrofacetDistribution ggx(alpha, alpha, 0.0f);

            bool entering = query.dirLocal.z >= 0.0f;
            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;
            Vector3D dirL = entering ? dirLocal : -dirLocal;

            Normal3D m = halfVector(dirL, dirV);
            float dotLH = dot(dirL, m);
            float commonPDFTerm = 1.0f / (4 * dotLH);

            float expectedF_D90 = 0.5f * m_roughness + 2 * m_roughness * query.dirLocal.z * query.dirLocal.z;
            float oneMinusDotVN5 = pow5(1 - dirV.z);
            float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
            float iBaseColor = m_diffuseColor.importance(query.wlHint) * expectedDiffuseFresnel * expectedDiffuseFresnel * lerp(1.0f, 1.0f / 1.51f, m_roughness);

            float expectedOneMinusDotVH5 = pow5(1 - dirV.z);
            float iSpecularF0 = m_specularF0Color.importance(query.wlHint);

            float diffuseWeight = iBaseColor;
            float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

            float sumWeights = diffuseWeight + specularWeight;

            float diffuseDirPDF = vlr::max(dirL.z / VLR_M_PI, 0.0f);
            float specularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirV, m);

            float ret = (diffuseDirPDF * diffuseWeight + specularDirPDF * specularWeight) / sumWeights;

            if (revValue) {
                float revDiffuseDirPDF = dirV.z / VLR_M_PI;
                float commonPDFTerm = 1.0f / (4 * dotLH);
                float revSpecularDirPDF = commonPDFTerm * ggx.evaluatePDF(dirL, m);

                float revExpectedF_D90 = 0.5f * m_roughness + 2 * m_roughness * pow2(dirL.z);
                float revOneMinusDotVN5 = pow5(1 - dirL.z);
                float revExpectedDiffuseFresnel = lerp(1.0f, revExpectedF_D90, revOneMinusDotVN5);
                float revIBaseColor = m_diffuseColor.importance(query.wlHint) * pow2(revExpectedDiffuseFresnel) * lerp(1.0f, 1.0f / 1.51f, m_roughness);

                float revExpectedOneMinusDotVH5 = pow5(1 - dirL.z);
                float revISpecularF0 = m_specularF0Color.importance(query.wlHint);

                float revDiffuseWeight = revIBaseColor;
                float revSpecularWeight = lerp(revISpecularF0, 1.0f, revExpectedOneMinusDotVH5);

                *revValue =
                    (revDiffuseDirPDF * revDiffuseWeight + revSpecularDirPDF * revSpecularWeight) /
                    (revDiffuseWeight + revSpecularWeight);
            }

            return ret;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            bool entering = query.dirLocal.z >= 0.0f;
            Vector3D dirV = entering ? query.dirLocal : -query.dirLocal;

            float expectedF_D90 = 0.5f * m_roughness + 2 * m_roughness * query.dirLocal.z * query.dirLocal.z;
            float oneMinusDotVN5 = pow5(1 - dirV.z);
            float expectedDiffuseFresnel = lerp(1.0f, expectedF_D90, oneMinusDotVN5);
            float iBaseColor = m_diffuseColor.importance(query.wlHint) * expectedDiffuseFresnel * expectedDiffuseFresnel * lerp(1.0f, 1.0f / 1.51f, m_roughness);

            float expectedOneMinusDotVH5 = pow5(1 - dirV.z);
            float iSpecularF0 = m_specularF0Color.importance(query.wlHint);

            float diffuseWeight = iBaseColor;
            float specularWeight = lerp(iSpecularF0, 1.0f, expectedOneMinusDotVH5);

            return diffuseWeight + specularWeight;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(UE4SurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<DiffuseAndSpecularBRDF*>(params);
        auto &mat = *reinterpret_cast<const UE4SurfaceMaterial*>(matDesc);

        SampledSpectrum baseColor = calcNode(mat.nodeBaseColor, mat.immBaseColor, surfPt, wls);
        float3 occlusionRoughnessMetallic = calcNode(mat.nodeOcclusionRoughnessMetallic,
                                                     make_float3(mat.immOcclusion, mat.immRoughness, mat.immMetallic),
                                                     surfPt, wls);
        float roughness = std::fmax(0.01f, occlusionRoughnessMetallic.y);
        float metallic = occlusionRoughnessMetallic.z;

        const float specular = 0.5f;
        SampledSpectrum diffuseColor = baseColor * (1 - metallic);
        SampledSpectrum specularF0Color = lerp(0.08f * specular * SampledSpectrum::One(), baseColor, metallic);

        p = DiffuseAndSpecularBRDF(diffuseColor, specularF0Color, roughness);

        return sizeof(DiffuseAndSpecularBRDF) / 4;
    }

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(OldStyleSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<DiffuseAndSpecularBRDF*>(params);
        auto &mat = *reinterpret_cast<const OldStyleSurfaceMaterial*>(matDesc);

        SampledSpectrum diffuseColor = calcNode(mat.nodeDiffuseColor, mat.immDiffuseColor, surfPt, wls);
        SampledSpectrum specularF0Color = calcNode(mat.nodeSpecularColor, mat.immSpecularColor, surfPt, wls);
        float roughness = std::fmax(0.01f, 1.0f - calcNode(mat.nodeGlossiness, mat.immGlossiness, surfPt, wls));

        p = DiffuseAndSpecularBRDF(diffuseColor, specularF0Color, roughness);

        return sizeof(DiffuseAndSpecularBRDF) / 4;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(DiffuseAndSpecularBRDF)



    class NullEDF {
    public:
        CUDA_DEVICE_FUNCTION NullEDF() {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            return false;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            return SampledSpectrum::Zero();
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            return 0.0f;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            return 0.0f;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(NullEDF_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        return 0;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(NullEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(NullEDF)



    class DiffuseEDF {
        SampledSpectrum m_emittance;

    public:
        CUDA_DEVICE_FUNCTION DiffuseEDF(const SampledSpectrum &emittance) :
            m_emittance(emittance) {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType = DirectionType::Emission() | DirectionType::LowFreq();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            result->dirLocal = cosineSampleHemisphere(uDir[0], uDir[1]);
            result->dirPDF = result->dirLocal.z / VLR_M_PI;
            result->sampledType = DirectionType::Emission() | DirectionType::LowFreq();
            SampledSpectrum feValue(1.0f / VLR_M_PI);

            return feValue;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            return m_emittance;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            return SampledSpectrum(dirLocal.z > 0.0f ? 1.0f / VLR_M_PI : 0.0f);
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            if (dirLocal.z <= 0.0f)
                return 0.0f;
            float pdfValue = dirLocal.z / VLR_M_PI;

            return pdfValue;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            return m_emittance.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(DiffuseEmitterSurfaceMaterial_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<DiffuseEDF*>(params);
        auto &mat = *reinterpret_cast<const DiffuseEmitterSurfaceMaterial*>(matDesc);

        p = DiffuseEDF(calcNode(mat.nodeEmittance, mat.immEmittance, surfPt, wls) * mat.immScale);

        return sizeof(DiffuseEDF) / 4;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(DiffuseEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(DiffuseEDF)



    class DirectionalEDF {
        SampledSpectrum m_emittance;
        Vector3D m_direction;

    public:
        CUDA_DEVICE_FUNCTION DirectionalEDF(const SampledSpectrum &emittance, const Vector3D &direction) :
            m_emittance(emittance), m_direction(direction) {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType;
            if constexpr (usePathSpaceRegularization)
                dirType = DirectionType::Emission() | DirectionType::HighFreq();
            else
                dirType = DirectionType::Emission() | DirectionType::Delta0D();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            float regFactor = 1.0f;
            DirectionType dirType = DirectionType::Delta0D();
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                regFactor = computeRegularizationFactor(&cosEpsilon);
                dirType = DirectionType::HighFreq();
            }

            result->dirLocal = Vector3D(0, 0, 1);
            result->dirPDF = regFactor;
            result->sampledType = DirectionType::Emission() | dirType;
            SampledSpectrum feValue(regFactor);

            return feValue;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            return m_emittance;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                if (dot(dirLocal, m_direction) >= cosEpsilon) {
                    SampledSpectrum ret = SampledSpectrum(regFactor);
                    return ret;
                }

                return SampledSpectrum::Zero();
            }
            else {
                return SampledSpectrum::Zero();
            }
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            if constexpr (usePathSpaceRegularization) {
                float cosEpsilon;
                float regFactor = computeRegularizationFactor(&cosEpsilon);

                if (dot(dirLocal, m_direction) >= cosEpsilon)
                    return regFactor;

                return 0.0f;
            }
            else {
                return 0.0f;
            }
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            return m_emittance.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(DirectionalEmitterSurfaceMaterial_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<DirectionalEDF*>(params);
        auto &mat = *reinterpret_cast<const DirectionalEmitterSurfaceMaterial*>(matDesc);

        p = DirectionalEDF(calcNode(mat.nodeEmittance, mat.immEmittance, surfPt, wls) * mat.immScale,
                           calcNode(mat.nodeDirection, mat.immDirection, surfPt, wls));

        return sizeof(DirectionalEDF) / 4;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(DirectionalEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(DirectionalEDF)



    class PointEDF {
        SampledSpectrum m_intensity;

    public:
        CUDA_DEVICE_FUNCTION PointEDF(const SampledSpectrum &intensity) :
            m_intensity(intensity) {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType = DirectionType::Emission() | DirectionType::LowFreq();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            result->dirLocal = uniformSampleSphere(uDir[0], uDir[1]);
            result->dirPDF = 1.0f / (4 * VLR_M_PI);
            result->sampledType = DirectionType::Emission() | DirectionType::LowFreq();

            return 1.0f / (4 * VLR_M_PI);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            // power = intensity * (4 * Pi)
            // area = 4 * pi * r^2
            // emittance = power / area = intensity / r^2
            return m_intensity;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            return 1.0f / (4 * VLR_M_PI);
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            float pdfValue = 1.0f / (4 * VLR_M_PI);

            return pdfValue;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            return m_intensity.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(PointEmitterSurfaceMaterial_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<PointEDF*>(params);
        auto &mat = *reinterpret_cast<const PointEmitterSurfaceMaterial*>(matDesc);

        p = PointEDF(calcNode(mat.nodeIntensity, mat.immIntensity, surfPt, wls) * mat.immScale);

        return sizeof(PointEDF) / 4;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(PointEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(PointEDF)



    // ----------------------------------------------------------------
    // MultiBSDF / MultiEDF

    // bsdf0-3: param offsets
    // numBSDFs
    // --------------------------------
    // BSDF0 procedure set index
    // BSDF0 params
    // ...
    // BSDF3 procedure set index
    // BSDF3 params
    class MultiBSDF {
        unsigned int m_bsdf0 : 6;
        unsigned int m_bsdf1 : 6;
        unsigned int m_bsdf2 : 6;
        unsigned int m_bsdf3 : 6;
        unsigned int m_numBSDFs : 8;

        CUDA_DEVICE_FUNCTION BSDFProcedureSet getBSDFProcSet(uint32_t offset, const uint32_t** body) const {
            const uint32_t* bsdf = reinterpret_cast<const uint32_t*>(this) + offset;
            uint32_t procIdx = *reinterpret_cast<const uint32_t*>(bsdf);
            *body = bsdf + 1;
            return plp.bsdfProcedureSetBuffer[procIdx];
        }

        CUDA_DEVICE_FUNCTION float BSDFWeight(uint32_t offset, const BSDFQuery &query) const {
            const uint32_t* body;
            BSDFProcedureSet procSet = getBSDFProcSet(offset, &body);
            auto matches = static_cast<ProgSigBSDFmatches>(procSet.progMatches);
            auto weightInternal = static_cast<ProgSigBSDFWeightInternal>(procSet.progWeightInternal);
            if (!matches(body, query.dirTypeFilter))
                return 0.0f;
            float weight_sn = weightInternal(body, query);
            float snCorrection;
            if (static_cast<TransportMode>(query.transportMode) == TransportMode::Radiance) {
                snCorrection = 1.0f;
            }
            else {
                snCorrection = std::fabs(query.dirLocal.z / dot(query.dirLocal, query.geometricNormalLocal));
                if (query.dirLocal.z == 0.0f)
                    snCorrection = 0.0f;
            }
            float ret = weight_sn * snCorrection;
            return ret;
        }

    public:
        CUDA_DEVICE_FUNCTION MultiBSDF(
            uint32_t bsdf0, uint32_t bsdf1, uint32_t bsdf2, uint32_t bsdf3,
            uint32_t numBSDFs) :
        m_bsdf0(bsdf0), m_bsdf1(bsdf1), m_bsdf2(bsdf2), m_bsdf3(bsdf3),
        m_numBSDFs(numBSDFs) {}

        CUDA_DEVICE_FUNCTION SampledSpectrum getBaseColor() const {
            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            SampledSpectrum ret;
            for (int i = 0; i < m_numBSDFs; ++i) {
                const uint32_t* bsdfBody;
                BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                auto getBaseColor = static_cast<ProgSigBSDFGetBaseColor>(procSet.progGetBaseColor);
                ret += getBaseColor(bsdfBody);
            }

            return ret;
        }

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            for (int i = 0; i < m_numBSDFs; ++i) {
                const uint32_t* bsdfBody;
                BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                auto matches = static_cast<ProgSigBSDFmatches>(procSet.progMatches);
                if (matches(bsdfBody, flags))
                    return true;
            }

            return false;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const BSDFQuery &query, float uComponent, const float uDir[2],
            BSDFQueryResult* result, BSDFQueryReverseResult* revResult = nullptr) const {
            if (revResult)
                revResult->dirPDF = 0.0f;

            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            float weights[4];
            for (int i = 0; i < m_numBSDFs; ++i)
                weights[i] = BSDFWeight(bsdfOffsets[i], query);

            // JP: 各BSDFのウェイトに基づいて方向のサンプルを行うBSDFを選択する。
            // EN: Based on the weight of each BSDF, select a BSDF from which direction sampling.
            float tempProb;
            float sumWeights;
            uint32_t idx = sampleDiscrete(weights, m_numBSDFs, uComponent, &tempProb, &sumWeights, &uComponent);
            if (sumWeights == 0.0f) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            // JP: 選択したBSDFから方向をサンプリングする。
            // EN: sample a direction from the selected BSDF.
            SampledSpectrum value;
            const uint32_t* selectedBsdfBody;
            BSDFProcedureSet selProcSet = getBSDFProcSet(bsdfOffsets[idx], &selectedBsdfBody);
            if (revResult) {
                auto sampleInternal = static_cast<ProgSigBSDFSampleWithRevInternal>(selProcSet.progSampleWithRevInternal);
                value = sampleInternal(selectedBsdfBody, query, uComponent, uDir, result, revResult);
            }
            else {
                auto sampleInternal = static_cast<ProgSigBSDFSampleInternal>(selProcSet.progSampleInternal);
                value = sampleInternal(selectedBsdfBody, query, uComponent, uDir, result);
            }

            // JP: 逆方向の確率密度を求めるための諸量を計算する。
            // EN: calculate quantities for reverse probability density.
            float revWeights[4];
            float sumRevWeights = 0;
            if (revResult) {
                BSDFQuery revQuery = query;// mQuery?
                Vector3D revDirIn = result->dirLocal;
                vlr::_swap(revQuery.dirLocal, revDirIn);
                revQuery.transportMode ^= 0b1;
                for (int i = 0; i < m_numBSDFs; ++i) {
                    revWeights[i] = BSDFWeight(bsdfOffsets[i], revQuery);
                    sumRevWeights += revWeights[i];
                }
            }

            result->dirPDF *= weights[idx];
            if (revResult)
                revResult->value *= revWeights[idx];
            if (result->dirPDF == 0.0f)
                return SampledSpectrum::Zero();

            // JP: サンプルした方向に関するBSDFの値の合計と、single-sample model MISに基づいた確率密度を計算する。
            // EN: calculate the total of BSDF values and a PDF based on the single-sample model MIS for the sampled direction.
            if (!result->sampledType.isDelta()) {
                for (int i = 0; i < m_numBSDFs; ++i) {
                    const uint32_t* bsdfBody;
                    BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                    if (revResult) {
                        auto evaluatePDFInternal = static_cast<ProgSigBSDFEvaluatePDFWithRevInternal>(procSet.progEvaluatePDFWithRevInternal);
                        if (i != idx && weights[i] > 0.0f) {
                            float revDirPDF;
                            float dirPDF = evaluatePDFInternal(bsdfBody, query, result->dirLocal, &revDirPDF);
                            result->dirPDF += dirPDF * weights[i];
                            revResult->dirPDF += revDirPDF * revWeights[i];
                        }
                    }
                    else {
                        auto evaluatePDFInternal = static_cast<ProgSigBSDFEvaluatePDFInternal>(procSet.progEvaluatePDFInternal);
                        if (i != idx && weights[i] > 0.0f)
                            result->dirPDF += evaluatePDFInternal(bsdfBody, query, result->dirLocal) * weights[i];
                    }
                }

                BSDFQuery mQuery = query;
                mQuery.dirTypeFilter &= sideTest(query.geometricNormalLocal, query.dirLocal, result->dirLocal);
                value = SampledSpectrum::Zero();
                if (revResult)
                    revResult->value = SampledSpectrum::Zero();
                for (int i = 0; i < m_numBSDFs; ++i) {
                    const uint32_t* bsdfBody;
                    BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                    if (revResult) {
                        auto evaluateInternal = static_cast<ProgSigBSDFEvaluateWithRevInternal>(procSet.progEvaluateWithRevInternal);
                        if (weights[i] == 0.0f)
                            continue;
                        SampledSpectrum revValue;
                        value += evaluateInternal(bsdfBody, mQuery, result->dirLocal, &revValue);
                        revResult->value += revValue;
                    }
                    else {
                        auto evaluateInternal = static_cast<ProgSigBSDFEvaluateInternal>(procSet.progEvaluateInternal);
                        if (weights[i] == 0.0f)
                            continue;
                        value += evaluateInternal(bsdfBody, mQuery, result->dirLocal);
                    }
                }
            }
            result->dirPDF /= sumWeights;
            if (revResult)
                revResult->dirPDF /= sumRevWeights;

            return value;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            SampledSpectrum* revValue = nullptr) const {
            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            SampledSpectrum retValue = SampledSpectrum::Zero();
            if (revValue)
                *revValue = SampledSpectrum::Zero();
            for (int i = 0; i < m_numBSDFs; ++i) {
                const uint32_t* bsdfBody;
                BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                auto matches = static_cast<ProgSigBSDFmatches>(procSet.progMatches);
                if (revValue) {
                    auto evaluateInternal = static_cast<ProgSigBSDFEvaluateWithRevInternal>(procSet.progEvaluateWithRevInternal);
                    if (!matches(bsdfBody, query.dirTypeFilter))
                        continue;
                    SampledSpectrum eRevValue;
                    retValue += evaluateInternal(bsdfBody, query, dirLocal, &eRevValue);
                    *revValue += eRevValue;
                }
                else {
                    auto evaluateInternal = static_cast<ProgSigBSDFEvaluateInternal>(procSet.progEvaluateInternal);
                    if (!matches(bsdfBody, query.dirTypeFilter))
                        continue;
                    retValue += evaluateInternal(bsdfBody, query, dirLocal);
                }
            }
            return retValue;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const BSDFQuery &query, const Vector3D &dirLocal,
            float* revValue = nullptr) const {
            if (revValue)
                *revValue = 0.0f;

            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            float sumWeights = 0.0f;
            float weights[4];
            for (int i = 0; i < m_numBSDFs; ++i) {
                weights[i] = BSDFWeight(bsdfOffsets[i], query);
                sumWeights += weights[i];
            }
            if (sumWeights == 0.0f)
                return 0.0f;

            float revWeights[4];
            float sumRevWeights = 0;
            if (revValue) {
                BSDFQuery revQuery = query;// mQuery?
                Vector3D revDirIn = dirLocal;
                vlr::_swap(revQuery.dirLocal, revDirIn);
                revQuery.transportMode ^= 0b1;
                for (int i = 0; i < m_numBSDFs; ++i) {
                    revWeights[i] = BSDFWeight(bsdfOffsets[i], revQuery);
                    sumRevWeights += revWeights[i];
                }
            }

            float retPDF = 0.0f;
            if (revValue)
                *revValue = 0.0f;
            for (int i = 0; i < m_numBSDFs; ++i) {
                const uint32_t* bsdfBody;
                BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                if (revValue) {
                    auto evaluatePDFInternal = static_cast<ProgSigBSDFEvaluatePDFWithRevInternal>(procSet.progEvaluatePDFWithRevInternal);
                    if (weights[i] > 0) {
                        float revDirPDF;
                        float dirPDF = evaluatePDFInternal(bsdfBody, query, dirLocal, &revDirPDF);
                        retPDF += dirPDF * weights[i];
                        *revValue += revDirPDF * revWeights[i];
                    }
                }
                else {
                    auto evaluatePDFInternal = static_cast<ProgSigBSDFEvaluatePDFInternal>(procSet.progEvaluatePDFInternal);
                    if (weights[i] > 0)
                        retPDF += evaluatePDFInternal(bsdfBody, query, dirLocal) * weights[i];
                }
            }
            retPDF /= sumWeights;
            if (revValue)
                *revValue /= sumRevWeights;

            return retPDF;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const BSDFQuery &query) const {
            uint32_t bsdfOffsets[4] = { m_bsdf0, m_bsdf1, m_bsdf2, m_bsdf3 };

            float ret = 0.0f;
            for (int i = 0; i < m_numBSDFs; ++i) {
                const uint32_t* bsdfBody;
                BSDFProcedureSet procSet = getBSDFProcSet(bsdfOffsets[i], &bsdfBody);
                auto matches = static_cast<ProgSigBSDFmatches>(procSet.progMatches);
                auto weightInternal = static_cast<ProgSigBSDFWeightInternal>(procSet.progWeightInternal);
                if (!matches(bsdfBody, query.dirTypeFilter))
                    continue;
                ret += weightInternal(bsdfBody, query);
            }

            return ret;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(MultiSurfaceMaterial_setupBSDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<MultiBSDF*>(params);
        auto &mat = *reinterpret_cast<const MultiSurfaceMaterial*>(matDesc);

        /*
        MultiBSDF
        ---- <-- MultiBSDF::bsdf0
        ProcedureSetIndex0
        Data0
        ---- <-- MultiBSDF::bsdf1
        ProcedureSetIndex1
        Data1
        ---- <-- MultiBSDF::bsdf2
        ProcedureSetIndex2
        Data2
        ---- <-- MultiBSDF::bsdf3
        ProcedureSetIndex3
        Data3
        */
        uint32_t baseIndex = sizeof(MultiBSDF) / 4;
        uint32_t bsdfOffsets[4] = { 0, 0, 0, 0 };
        for (int i = 0; i < mat.numSubMaterials; ++i) {
            bsdfOffsets[i] = baseIndex;

            const SurfaceMaterialDescriptor subMatDesc = plp.materialDescriptorBuffer[mat.subMatIndices[i]];
            auto setupBSDF = static_cast<ProgSigSetupBSDF>(subMatDesc.progSetupBSDF);
            *(params + baseIndex++) = subMatDesc.bsdfProcedureSetIndex;
            baseIndex += setupBSDF(subMatDesc.data, surfPt, wls, params + baseIndex);
        }

        p = MultiBSDF(
            bsdfOffsets[0],
            bsdfOffsets[1],
            bsdfOffsets[2],
            bsdfOffsets[3],
            mat.numSubMaterials);

        //vlrDevPrintf("%u, %u, %u, %u, %u mats\n", p.bsdf0, p.bsdf1, p.bsdf2, p.bsdf3, p.numBSDFs);

        return baseIndex;
    }

    DEFINE_BSDF_CALLABLE_PROGRAMS(MultiBSDF)

    // edf0-3: param offsets
    // numEDFs
    // --------------------------------
    // EDF0 procedure set index
    // EDF0 params
    // ...
    // EDF3 procedure set index
    // EDF3 params
    class MultiEDF {
        unsigned int m_edf0 : 6;
        unsigned int m_edf1 : 6;
        unsigned int m_edf2 : 6;
        unsigned int m_edf3 : 6;
        unsigned int m_numEDFs : 8;

        CUDA_DEVICE_FUNCTION const uint32_t* getEDF(uint32_t offset) const {
            return reinterpret_cast<const uint32_t*>(this) + offset;
        }

    public:
        CUDA_DEVICE_FUNCTION MultiEDF(
            uint32_t edf0, uint32_t edf1, uint32_t edf2, uint32_t edf3,
            uint32_t numEDFs) :
            m_edf0(edf0), m_edf1(edf1), m_edf2(edf2), m_edf3(edf3),
            m_numEDFs(numEDFs) {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto matches = static_cast<ProgSigEDFmatches>(procSet.progMatches);

                if (matches(edf + 1, flags))
                    return true;
            }

            return false;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            float weights[4];
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto weightInternal = static_cast<ProgSigEDFWeightInternal>(procSet.progWeightInternal);

                weights[i] = weightInternal(edf + 1, query);
            }

            // JP: 各EDFのウェイトに基づいて方向のサンプルを行うEDFを選択する。
            // EN: Based on the weight of each EDF, select a EDF from which direction sampling.
            float tempProb;
            float sumWeights;
            uint32_t idx = sampleDiscrete(weights, m_numEDFs, uComponent, &tempProb, &sumWeights, &uComponent);
            if (sumWeights == 0.0f) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            const uint32_t* selectedEDF = getEDF(edfOffsets[idx]);
            uint32_t selProcIdx = *reinterpret_cast<const uint32_t*>(selectedEDF);
            const EDFProcedureSet selProcSet = plp.edfProcedureSetBuffer[selProcIdx];
            auto sampleInternal = static_cast<ProgSigEDFSampleInternal>(selProcSet.progSampleInternal);

            // JP: 選択したEDFから方向をサンプリングする。
            // EN: sample a direction from the selected EDF.
            SampledSpectrum value = sampleInternal(selectedEDF + 1, query, uComponent, uDir, result);
            result->dirPDF *= weights[idx];
            if (result->dirPDF == 0.0f) {
                result->dirPDF = 0.0f;
                return SampledSpectrum::Zero();
            }

            // JP: サンプルした方向に関するEDFの値の合計と、single-sample model MISに基づいた確率密度を計算する。
            // EN: calculate the total of EDF values and a PDF based on the single-sample model MIS for the sampled direction.
            if (!result->sampledType.isDelta()) {
                for (int i = 0; i < m_numEDFs; ++i) {
                    const uint32_t* edf = getEDF(edfOffsets[i]);
                    uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                    const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                    auto matches = static_cast<ProgSigEDFmatches>(procSet.progMatches);
                    auto evaluatePDFInternal = static_cast<ProgSigEDFEvaluatePDFInternal>(procSet.progEvaluatePDFInternal);

                    if (i != idx && matches(edf + 1, query.dirTypeFilter))
                        result->dirPDF += evaluatePDFInternal(edf + 1, query, result->dirLocal) * weights[i];
                }

                value = SampledSpectrum::Zero();
                for (int i = 0; i < m_numEDFs; ++i) {
                    const uint32_t* edf = getEDF(edfOffsets[i]);
                    uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                    const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                    auto matches = static_cast<ProgSigEDFmatches>(procSet.progMatches);
                    auto evaluateInternal = static_cast<ProgSigEDFEvaluateInternal>(procSet.progEvaluateInternal);

                    if (!matches(edf + 1, query.dirTypeFilter))
                        continue;
                    value += evaluateInternal(edf + 1, query, result->dirLocal);
                }
            }
            result->dirPDF /= sumWeights;

            return value;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            SampledSpectrum ret = SampledSpectrum::Zero();
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto evaluateEmittanceInternal = static_cast<ProgSigEDFEvaluateEmittanceInternal>(procSet.progEvaluateEmittanceInternal);

                ret += evaluateEmittanceInternal(edf + 1);
            }

            return ret;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            SampledSpectrum ret = SampledSpectrum::Zero();
            SampledSpectrum sumEmittance = SampledSpectrum::Zero();
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto evaluateEmittanceInternal = static_cast<ProgSigEDFEvaluateEmittanceInternal>(procSet.progEvaluateEmittanceInternal);
                auto evaluateInternal = static_cast<ProgSigEDFEvaluateInternal>(procSet.progEvaluateInternal);

                SampledSpectrum emittance = evaluateEmittanceInternal(edf + 1);
                sumEmittance += emittance;
                ret += emittance * evaluateInternal(edf + 1, query, dirLocal);
            }
            ret.safeDivide(sumEmittance);

            return ret;
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            float sumWeights = 0.0f;
            float weights[4];
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);;
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto weightInternal = static_cast<ProgSigEDFWeightInternal>(procSet.progWeightInternal);

                weights[i] = weightInternal(edf + 1, query);
                sumWeights += weights[i];
            }
            if (sumWeights == 0.0f)
                return 0.0f;

            float retPDF = 0.0f;
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);;
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto evaluatePDFInternal = static_cast<ProgSigEDFEvaluatePDFInternal>(procSet.progEvaluatePDFInternal);

                if (weights[i] > 0)
                    retPDF += evaluatePDFInternal(edf + 1, query, dirLocal) * weights[i];
            }
            retPDF /= sumWeights;

            return retPDF;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            uint32_t edfOffsets[4] = { m_edf0, m_edf1, m_edf2, m_edf3 };

            float ret = 0.0f;
            for (int i = 0; i < m_numEDFs; ++i) {
                const uint32_t* edf = getEDF(edfOffsets[i]);
                uint32_t procIdx = *reinterpret_cast<const uint32_t*>(edf);;
                const EDFProcedureSet procSet = plp.edfProcedureSetBuffer[procIdx];
                auto weightInternal = static_cast<ProgSigEDFWeightInternal>(procSet.progWeightInternal);

                ret += weightInternal(edf + 1, query);
            }

            return ret;
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(MultiSurfaceMaterial_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<MultiEDF*>(params);
        auto &mat = *reinterpret_cast<const MultiSurfaceMaterial*>(matDesc);

        uint32_t baseIndex = sizeof(MultiEDF) / 4;
        uint32_t edfOffsets[4] = { 0, 0, 0, 0 };
        for (int i = 0; i < mat.numSubMaterials; ++i) {
            edfOffsets[i] = baseIndex;

            const SurfaceMaterialDescriptor subMatDesc = plp.materialDescriptorBuffer[mat.subMatIndices[i]];
            ProgSigSetupEDF setupEDF = (ProgSigSetupEDF)subMatDesc.progSetupEDF;
            *(params + baseIndex++) = subMatDesc.edfProcedureSetIndex;
            baseIndex += setupEDF(subMatDesc.data, surfPt, wls, params + baseIndex);
        }

        p = MultiEDF(
            edfOffsets[0],
            edfOffsets[1],
            edfOffsets[2],
            edfOffsets[3],
            mat.numSubMaterials);

        return baseIndex;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(MultiEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(MultiEDF)

    // END: MultiBSDF / MultiEDF
    // ----------------------------------------------------------------



    class EnvironmentEDF {
        SampledSpectrum m_emittance;
        float m_worldRadius;
        float m_worldDiscArea;

    public:
        CUDA_DEVICE_FUNCTION EnvironmentEDF(
            const SampledSpectrum &emittance, float worldRadius, float worldDiscArea) :
            m_emittance(emittance), m_worldRadius(worldRadius), m_worldDiscArea(worldDiscArea) {}

        CUDA_DEVICE_FUNCTION bool matches(DirectionType flags) const {
            DirectionType dirType = DirectionType::Emission() | DirectionType::LowFreq();
            return dirType.matches(flags);
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum sampleInternal(
            const EDFQuery &query, float uComponent, const float uDir[2], EDFQueryResult* result) const {
            float dx, dy;
            concentricSampleDisk(uDir[0], uDir[1], &dx, &dy);

            // encode the projected position as the xy components.
            result->dirLocal = Vector3D(m_worldRadius * dx, m_worldRadius * dy, 1);
            // The true value is: lim_{l to inf} l^2 / m_worldDiscArea
            result->dirPDF = 1.0f / m_worldDiscArea;
            result->sampledType = DirectionType::Emission() | DirectionType::LowFreq();
            SampledSpectrum feValue(1 / VLR_M_PI);

            return feValue;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateEmittanceInternal() const {
            return VLR_M_PI * m_emittance;
        }

        CUDA_DEVICE_FUNCTION SampledSpectrum evaluateInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            return SampledSpectrum(dirLocal.z > 0.0f ? 1.0f / VLR_M_PI : 0.0f);
        }

        CUDA_DEVICE_FUNCTION float evaluatePDFInternal(
            const EDFQuery &query, const Vector3D &dirLocal) const {
            if (dirLocal.z <= 0.0f)
                return 0.0f;
            // The true value is: lim_{l to inf} l^2 / m_worldDiscArea
            float pdfValue = 1.0f / m_worldDiscArea;

            return pdfValue;
        }

        CUDA_DEVICE_FUNCTION float weightInternal(const EDFQuery &query) const {
            return m_emittance.importance(query.wlHint);
        }
    };

    RT_CALLABLE_PROGRAM uint32_t RT_DC_NAME(EnvironmentEmitterSurfaceMaterial_setupEDF)(
        const uint32_t* matDesc, const SurfacePoint &surfPt, const WavelengthSamples &wls, uint32_t* params) {
        auto &p = *reinterpret_cast<EnvironmentEDF*>(params);
        auto &mat = *reinterpret_cast<const EnvironmentEmitterSurfaceMaterial*>(matDesc);

        p = EnvironmentEDF(calcNode(mat.nodeEmittance, mat.immEmittance, surfPt, wls) * mat.immScale,
                           plp.sceneBounds->worldRadius, plp.sceneBounds->worldDiscArea);

        return sizeof(EnvironmentEDF) / 4;
    }

    DEFINE_EDF_CALLABLE_PROGRAMS(EnvironmentEDF)
    DEFINE_EDF_AS_BSDF_CALLABLE_PROGRAMS(EnvironmentEDF)
}
