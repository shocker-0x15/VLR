#include "hip/hip_runtime.h"
﻿#define RT_PIPELINE_LAUNCH_PARAMETERS extern "C" __constant__
#include "../shared/shared.h"

namespace vlr {
    using namespace shared;

    CUDA_DEVICE_KERNEL void convertToRGB(const optixu::BlockBuffer2D<SpectrumStorage, 0> spectrumBuffer,
                                         optixu::NativeBlockBuffer2D<float4> rgbBuffer,
                                         uint32_t numAccumFrames) {
        uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                       blockDim.y * blockIdx.y + threadIdx.y);
        uint2 imageSize = spectrumBuffer.getSize();
        if (launchIndex.x >= imageSize.x || launchIndex.y >= imageSize.y)
            return;
        const DiscretizedSpectrum &spectrum = spectrumBuffer[launchIndex].getValue().result;
        float XYZ[3];
        spectrum.toXYZ(XYZ);
        //if (launchIndex.x == 640 && launchIndex.y == 128) {
        //    spectrum.print();
        //    vlrprintf("%g, %g, %g\n", XYZ[0], XYZ[1], XYZ[2]);
        //}
        VLRAssert(XYZ[0] >= 0.0f && XYZ[1] >= 0.0f && XYZ[2] >= 0.0f, "each value of XYZ must not be negative.");
        float recNumAccums = 1.0f / numAccumFrames;
        XYZ[0] *= recNumAccums;
        XYZ[1] *= recNumAccums;
        XYZ[2] *= recNumAccums;
        //pv_RGBBuffer[sm_launchIndex] = RGBSpectrum(XYZ[0], XYZ[1], XYZ[2]);
        float RGB[3];
        transformTristimulus(mat_XYZ_to_Rec709_D65, XYZ, RGB);
        rgbBuffer.write(launchIndex, make_float4(RGB[0], RGB[1], RGB[2], 1.0f)); // not clamp out of gamut color.
    }
}
