#include "hip/hip_runtime.h"
#include "../shared/light_transport_common.h"

// Reference
// Progressive Light Transport Simulation on the GPU: Survey and Improvements

namespace vlr {
    using namespace shared;

    CUDA_DEVICE_KERNEL void RT_AH_NAME(lvcbptAnyHitWithAlpha)() {
        LTReadOnlyPayload* roPayload;
        LTReadWritePayload* rwPayload;
        LTPayloadSignature::get(&roPayload, nullptr, &rwPayload);

        float alpha = getAlpha(roPayload->wls);

        // Stochastic Alpha Test
        if (rwPayload->rng.getFloat0cTo1o() >= alpha)
            optixIgnoreIntersection();
    }

    CUDA_DEVICE_FUNCTION void atomicAddToBuffer(
        const WavelengthSamples &wls, SampledSpectrum contribution,
        const float2 &pixel) {
        uint32_t ipx = static_cast<uint32_t>(pixel.x);
        uint32_t ipy = static_cast<uint32_t>(pixel.y);
        if (ipx < plp.imageSize.x && ipy < plp.imageSize.y) {
            if (!contribution.allFinite()) {
                vlrprintf("Pass %u, (%u - %u, %u): Not a finite value.\n",
                          plp.numAccumFrames, optixGetLaunchIndex().x, ipx, ipy);
                return;
            }
            plp.atomicAccumBuffer[ipy * plp.imageStrideInPixels + ipx].atomicAdd(wls, contribution);
        }
    }

    CUDA_DEVICE_FUNCTION void storeLightVertex(
        float totalPowerProbDensity, float prevTotalPowerProbDensity, float prevSumPowerProbDensities,
        float backwardConversionFactor,
        const SampledSpectrum &flux, const Vector3D &dirInLocal, DirectionType sampledType, bool wlSelected,
        const SurfacePoint &surfPt, uint32_t pathLength) {
        LightPathVertex lightVertex = {};
        lightVertex.instIndex = surfPt.instIndex;
        lightVertex.geomInstIndex = surfPt.geomInstIndex;
        lightVertex.primIndex = surfPt.primIndex;
        lightVertex.u = surfPt.u;
        lightVertex.v = surfPt.v;
        lightVertex.totalPowerProbDensity = totalPowerProbDensity;
        lightVertex.prevTotalPowerProbDensity = prevTotalPowerProbDensity;
        lightVertex.prevSumPowerProbDensities = prevSumPowerProbDensities;
        lightVertex.backwardConversionFactor = backwardConversionFactor;
        lightVertex.flux = flux;
        lightVertex.dirInLocal = dirInLocal;
        lightVertex.sampledType = sampledType;
        lightVertex.wlSelected = wlSelected;
        lightVertex.pathLength = pathLength;
        uint32_t cacheIndex = atomicAdd(plp.numLightVertices, 1u);
        plp.lightVertexCache[cacheIndex] = lightVertex;
    }



    CUDA_DEVICE_KERNEL void RT_RG_NAME(lvcbptLightPath)() {
        uint32_t launchIndex = optixGetLaunchIndex().x;

        KernelRNG rng = plp.linearRngBuffer[launchIndex];

        float uLight = rng.getFloat0cTo1o();
        SurfaceLight light;
        float lightProb;
        float uPrim;
        selectSurfaceLight(uLight, &light, &lightProb, &uPrim);

        SurfaceLightPosSample Le0Sample(uPrim, rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        SurfaceLightPosQueryResult Le0Result;
        light.sample(Le0Sample, Point3D(NAN), &Le0Result);

        const SurfaceMaterialDescriptor lightMatDesc = plp.materialDescriptorBuffer[Le0Result.materialIndex];
        EDF edf(lightMatDesc, Le0Result.surfPt, plp.commonWavelengthSamples);

        float probDensity0 = plp.numLightPaths * lightProb * Le0Result.areaPDF;
        SampledSpectrum Le0 = edf.evaluateEmittance();
        SampledSpectrum alpha = Le0 / probDensity0;

        float powerProbDensities0 = pow2(probDensity0);
        float prevPowerProbDensity0 = pow2(1);
        float prevSumPowerProbDensities0 = 0;
        storeLightVertex(powerProbDensities0, prevPowerProbDensity0, prevSumPowerProbDensities0, 0,
                         alpha, Vector3D(0, 0, 1), Le0Result.posType, false,
                         Le0Result.surfPt, 0);

        EDFQuery edfQuery(DirectionType::All(), plp.commonWavelengthSamples);
        EDFSample Le1Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        EDFQueryResult Le1Result;
        SampledSpectrum Le1 = edf.sample(edfQuery, Le1Sample, &Le1Result);

        Point3D rayOrg = offsetRayOrigin(Le0Result.surfPt.position, Le0Result.surfPt.geometricNormal);
        if (Le0Result.surfPt.atInfinity) {
            rayOrg = plp.sceneBounds->center +
                1.1f * plp.sceneBounds->worldRadius * Le0Result.surfPt.position +
                Le0Result.surfPt.shadingFrame.x * Le1Result.dirLocal.x +
                Le0Result.surfPt.shadingFrame.y * Le1Result.dirLocal.y;
            Le1Result.dirLocal.x = 0;
            Le1Result.dirLocal.y = 0;
        }
        Vector3D rayDir = Le0Result.surfPt.fromLocal(Le1Result.dirLocal);
        float cosTerm = Le0Result.surfPt.calcCosTerm(rayDir);
        alpha *= Le1 * (cosTerm / Le1Result.dirPDF);

        LVCBPTLightPathReadOnlyPayload roPayload = {};
        roPayload.prevDirPDF = Le1Result.dirPDF;
        roPayload.prevCosTerm = cosTerm;
        roPayload.prevRevAreaPDF = 0;
        roPayload.prevSampledType = Le1Result.sampledType;
        LVCBPTLightPathWriteOnlyPayload woPayload = {};
        LVCBPTLightPathReadWritePayload rwPayload = {};
        rwPayload.rng = rng;
        rwPayload.alpha = alpha;
        rwPayload.totalPowerProbDensity = powerProbDensities0;
        rwPayload.prevTotalPowerProbDensity = prevPowerProbDensity0;
        rwPayload.prevSumPowerProbDensities = prevSumPowerProbDensities0;
        rwPayload.singleIsSelected = false;
        rwPayload.pathLength = 0;
        LVCBPTLightPathReadOnlyPayload* roPayloadPtr = &roPayload;
        LVCBPTLightPathWriteOnlyPayload* woPayloadPtr = &woPayload;
        LVCBPTLightPathReadWritePayload* rwPayloadPtr = &rwPayload;

        const uint32_t MaxPathLength = 25;
        while (true) {
            rwPayload.terminate = true;
            ++rwPayload.pathLength;

            optixu::trace<LVCBPTLightPathPayloadSignature>(
                plp.topGroup, asOptiXType(rayOrg), asOptiXType(rayDir), 0.0f, FLT_MAX, 0.0f,
                shared::VisibilityGroup_Everything, OPTIX_RAY_FLAG_NONE,
                LVCBPTRayType::LightPath, MaxNumRayTypes, LVCBPTRayType::LightPath,
                roPayloadPtr, woPayloadPtr, rwPayloadPtr);

            if (rwPayload.pathLength >= MaxPathLength)
                rwPayload.terminate = true;
            if (rwPayload.terminate)
                break;

            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
            roPayload.prevDirPDF = woPayload.dirPDF;
            roPayload.prevCosTerm = woPayload.cosTerm;
            roPayload.prevRevAreaPDF = woPayload.revAreaPDF;
            roPayload.prevSampledType = woPayload.sampledType;
        }
        plp.linearRngBuffer[launchIndex] = rwPayload.rng;
    }



    CUDA_DEVICE_KERNEL void RT_CH_NAME(lvcbptLightPath)() {
        const auto hp = HitPointParameter::get();

        LVCBPTLightPathReadOnlyPayload* roPayload;
        LVCBPTLightPathWriteOnlyPayload* woPayload;
        LVCBPTLightPathReadWritePayload* rwPayload;
        LVCBPTLightPathPayloadSignature::get(&roPayload, &woPayload, &rwPayload);

        KernelRNG &rng = rwPayload->rng;
        WavelengthSamples wls = plp.commonWavelengthSamples;

        SurfacePoint surfPt;
        float hypAreaPDF;
        calcSurfacePoint(hp, wls, &surfPt, &hypAreaPDF);

        const SurfaceMaterialDescriptor matDesc = plp.materialDescriptorBuffer[hp.sbtr->geomInst.materialIndex];
        constexpr TransportMode transportMode = TransportMode::Importance;
        BSDF<transportMode, BSDFTier::Bidirectional> bsdf(matDesc, surfPt, wls);

        Vector3D dirIn = -asVector3D(optixGetWorldRayDirection());
        Vector3D dirInLocal = surfPt.shadingFrame.toLocal(dirIn);

        Normal3D geomNormalLocal = surfPt.shadingFrame.toLocal(surfPt.geometricNormal);
        BSDFQuery fsQuery(dirInLocal, geomNormalLocal, transportMode, DirectionType::All(), wls);

        rwPayload->prevSumPowerProbDensities =
            rwPayload->prevTotalPowerProbDensity +
            pow2(roPayload->prevRevAreaPDF) * rwPayload->prevSumPowerProbDensities;
        rwPayload->prevTotalPowerProbDensity = rwPayload->totalPowerProbDensity;

        float lastDist2 = sqDistance(asPoint3D(optixGetWorldRayOrigin()), surfPt.position);
        float probDensity = roPayload->prevDirPDF * absDot(dirInLocal, geomNormalLocal) / lastDist2;
        //if (!vlr::isinf(rwPayload->totalPowerProbDensity) &&
        //    vlr::isinf(rwPayload->totalPowerProbDensity * pow2(probDensity))) {
        //    printf("LightPath: %g, %g\n", rwPayload->totalPowerProbDensity, pow2(probDensity));
        //}
        rwPayload->totalPowerProbDensity *= pow2(probDensity);

        storeLightVertex(rwPayload->totalPowerProbDensity,
                         rwPayload->prevTotalPowerProbDensity, rwPayload->prevSumPowerProbDensities,
                         roPayload->prevCosTerm / lastDist2,
                         rwPayload->alpha, dirInLocal,
                         roPayload->prevSampledType, rwPayload->singleIsSelected, surfPt, rwPayload->pathLength);

        BSDFSample sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        BSDFQueryResult fsResult;
        BSDFQueryReverseResult fsRevResult;
        SampledSpectrum fs = bsdf.sample(fsQuery, sample, &fsResult, &fsRevResult);
        if (fs == SampledSpectrum::Zero() || fsResult.dirPDF == 0.0f)
            return;
        if (fsResult.sampledType.isDispersive() && !wls.singleIsSelected())
            rwPayload->singleIsSelected = true;

        float cosTerm = dot(fsResult.dirLocal, geomNormalLocal);
        SampledSpectrum throughput = fs * (std::fabs(cosTerm) / fsResult.dirPDF);
        rwPayload->alpha *= throughput;

        // Russian roulette
        float continueProb = std::fmin(throughput.importance(wls.selectedLambdaIndex()), 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb)
            return;
        rwPayload->alpha /= continueProb;
        rwPayload->terminate = false;

        Vector3D dirOut = surfPt.fromLocal(fsResult.dirLocal);
        woPayload->nextOrigin = offsetRayOrigin(surfPt.position, cosTerm > 0.0f ? surfPt.geometricNormal : -surfPt.geometricNormal);
        woPayload->nextDirection = dirOut;
        woPayload->dirPDF = fsResult.dirPDF;
        woPayload->cosTerm = std::fabs(cosTerm);
        woPayload->revAreaPDF = fsRevResult.dirPDF * roPayload->prevCosTerm / lastDist2;
        woPayload->sampledType = fsResult.sampledType;
    }



    static constexpr int32_t debugPathLength = 0;

    CUDA_DEVICE_KERNEL void RT_RG_NAME(lvcbptEyePath)() {
        uint2 launchIndex = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);

        KernelRNG rng = plp.rngBuffer.read(launchIndex);

        float2 p = make_float2(launchIndex.x + rng.getFloat0cTo1o(),
                               launchIndex.y + rng.getFloat0cTo1o());

        float resCorrection = plp.imageSize.x * plp.imageSize.y;
        WavelengthSamples wls = plp.commonWavelengthSamples;

        Camera camera(static_cast<ProgSigCamera_sample>(plp.progSampleLensPosition));
        LensPosSample We0Sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        LensPosQueryResult We0Result;
        camera.sample(We0Sample, &We0Result);

        IDF idf(plp.cameraDescriptor, We0Result.surfPt, wls);

        SampledSpectrum We0 = idf.evaluateSpatialImportance();
        SampledSpectrum alpha = We0 / We0Result.areaPDF;

        float powerProbDensities0 = pow2(We0Result.areaPDF);
        float prevPowerProbDensity0 = pow2(1);
        float prevSumPowerProbDensities0 = 0;

        IDFQuery idfQuery;

        // Connect with a randomly chosen light vertex.
        {
            const SurfacePoint &surfPtE = We0Result.surfPt;
            Normal3D geomNormalLocalE = surfPtE.shadingFrame.toLocal(surfPtE.geometricNormal);

            uint32_t lightVertexIndex = vlr::min<uint32_t>(
                *plp.numLightVertices * rng.getFloat0cTo1o(),
                *plp.numLightVertices - 1);
            const LightPathVertex &vertex = plp.lightVertexCache[lightVertexIndex];
            float vertexProb = 1.0f / *plp.numLightVertices;

            SurfacePoint surfPtL;
            uint32_t matIndexL;
            {
                const GeometryInstance &geomInst = plp.geomInstBuffer[vertex.geomInstIndex];
                ProgSigDecodeHitPoint decodeHitPoint(geomInst.progDecodeHitPoint);
                decodeHitPoint(vertex.instIndex, vertex.geomInstIndex, vertex.primIndex,
                               vertex.u, vertex.v, &surfPtL);

                Normal3D localNormal = calcNode(geomInst.nodeNormal, Normal3D(0.0f, 0.0f, 1.0f), surfPtL, wls);
                applyBumpMapping(localNormal, &surfPtL);

                Vector3D newTangent = calcNode(geomInst.nodeTangent, surfPtL.shadingFrame.x, surfPtL, wls);
                modifyTangent(newTangent, &surfPtL);

                matIndexL = geomInst.materialIndex;
            }

            Vector3D conRayDir;
            float squaredConDist;
            float fractionalVisibility;
            if ((debugPathLength == 0 || (vertex.pathLength + 1) == debugPathLength) &&
                testVisibility<LVCBPTRayType::Connection>(
                    surfPtE, surfPtL, wls, &conRayDir, &squaredConDist, &fractionalVisibility)) {
                float recSquaredConDist = 1.0f / squaredConDist;

                const SurfaceMaterialDescriptor matDescL = plp.materialDescriptorBuffer[matIndexL];
                constexpr TransportMode transportModeL = TransportMode::Importance;
                BSDF<transportModeL, BSDFTier::Bidirectional> bsdfL(matDescL, surfPtL, wls, vertex.pathLength == 0);
                Vector3D dirInLocalL = vertex.dirInLocal;
                Normal3D geomNormalLocalL = surfPtL.shadingFrame.toLocal(surfPtL.geometricNormal);
                BSDFQuery bsdfLQuery(dirInLocalL, geomNormalLocalL, transportModeL, DirectionType::All(), wls);

                Vector3D conRayDirLocalL = surfPtL.toLocal(-conRayDir);
                Vector3D conRayDirLocalE = surfPtE.toLocal(conRayDir);

                float cosL = absDot(conRayDirLocalL, geomNormalLocalL);
                float cosE = absDot(conRayDirLocalE, geomNormalLocalE);
                float G = cosL * cosE * recSquaredConDist;

                SampledSpectrum backwardFsL;
                SampledSpectrum forwardFsL = bsdfL.evaluate(bsdfLQuery, conRayDirLocalL, &backwardFsL);
                float backwardDirDensityL;
                /*float forwardDirDensityL = */bsdfL.evaluatePDF(bsdfLQuery, conRayDirLocalL, &backwardDirDensityL);
                //float forwardAreaDensityL = forwardDirDensityL * cosE * recSquaredConDist;
                float backwardAreaDensityL = backwardDirDensityL * vertex.backwardConversionFactor;
                float partialDenomMisWeightL = vertex.prevTotalPowerProbDensity +
                    pow2(backwardAreaDensityL) * vertex.prevSumPowerProbDensities; // extend eye subpath, shorten light subpath.

                SampledSpectrum backwardFsE;
                SampledSpectrum forwardFsE = idf.evaluateDirectionalImportance(idfQuery, conRayDirLocalE);
                float forwardDirDensityE = idf.evaluatePDF(idfQuery, conRayDirLocalE);
                forwardDirDensityE *= resCorrection;
                float forwardAreaDensityE = forwardDirDensityE * cosL * recSquaredConDist;
                float2 posInScreen = idf.backProjectDirection(idfQuery, conRayDirLocalE);
                float2 pixel = make_float2(posInScreen.x * plp.imageSize.x, posInScreen.y * plp.imageSize.y);

                // JP: ���C�g�g���[�V���O���s�N�Z�������s���邱�Ƃɓ������̂Ŋm�����x���s�N�Z�����{�ɂȂ�B
                float scalarTerm = G * fractionalVisibility /
                    (vertexProb * resCorrection * plp.wavelengthProbability);
                if (vertex.wlSelected)
                    scalarTerm *= SampledSpectrum::NumComponents();
                SampledSpectrum conTerm = forwardFsL * scalarTerm * forwardFsE;
                SampledSpectrum unweightedContribution = vertex.flux * conTerm * alpha;

                float numMisWeight = powerProbDensities0 * pow2(resCorrection) * vertex.totalPowerProbDensity;
                float denomMisWeight = numMisWeight;
                // extend eye subpath, shorten light subpath.
                denomMisWeight += powerProbDensities0 * pow2(forwardAreaDensityE) * partialDenomMisWeightL;

                float misWeight = 10 * numMisWeight / denomMisWeight;
                SampledSpectrum contribution = misWeight * unweightedContribution;
                atomicAddToBuffer(wls, contribution, pixel);
            }
        }

        IDFSample We1Sample(p.x / plp.imageSize.x, p.y / plp.imageSize.y);
        IDFQueryResult We1Result;
        SampledSpectrum We1 = idf.sample(idfQuery, We1Sample, &We1Result);
        We1Result.dirPDF *= resCorrection;

        Point3D rayOrg = We0Result.surfPt.position;
        Vector3D rayDir = We0Result.surfPt.fromLocal(We1Result.dirLocal);
        float cosTerm = We0Result.surfPt.calcCosTerm(rayDir);
        alpha *= We1 * (cosTerm / We1Result.dirPDF);

        LVCBPTEyePathReadOnlyPayload roPayload = {};
        roPayload.prevDirPDF = We1Result.dirPDF;
        roPayload.prevCosTerm = cosTerm;
        roPayload.prevRevAreaPDF = 0;
        roPayload.prevSampledType = We1Result.sampledType;
        LVCBPTEyePathWriteOnlyPayload woPayload = {};
        LVCBPTEyePathReadWritePayload rwPayload = {};
        rwPayload.rng = rng;
        rwPayload.alpha = alpha;
        rwPayload.contribution = SampledSpectrum::Zero();
        rwPayload.totalPowerProbDensity = powerProbDensities0;
        rwPayload.prevTotalPowerProbDensity = prevPowerProbDensity0;
        rwPayload.prevSumPowerProbDensities = prevSumPowerProbDensities0;
        rwPayload.singleIsSelected = false;
        rwPayload.pathLength = 0;
        LVCBPTEyePathExtraPayload exPayload = {};
        LVCBPTEyePathReadOnlyPayload* roPayloadPtr = &roPayload;
        LVCBPTEyePathWriteOnlyPayload* woPayloadPtr = &woPayload;
        LVCBPTEyePathReadWritePayload* rwPayloadPtr = &rwPayload;
        LVCBPTEyePathExtraPayload* exPayloadPtr = &exPayload;

        const uint32_t MaxPathLength = 25;
        while (true) {
            rwPayload.terminate = true;
            ++rwPayload.pathLength;

            if (debugPathLength != 0 &&
                rwPayload.pathLength > debugPathLength)
                break;

            optixu::trace<LVCBPTEyePathPayloadSignature>(
                plp.topGroup, asOptiXType(rayOrg), asOptiXType(rayDir), 0.0f, FLT_MAX, 0.0f,
                shared::VisibilityGroup_Everything, OPTIX_RAY_FLAG_NONE,
                LVCBPTRayType::EyePath, MaxNumRayTypes, LVCBPTRayType::EyePath,
                roPayloadPtr, woPayloadPtr, rwPayloadPtr, exPayloadPtr);

            if (rwPayload.pathLength == 1) {
                uint32_t linearIndex = launchIndex.y * plp.imageStrideInPixels + launchIndex.x;
                DiscretizedSpectrum &accumAlbedo = plp.accumAlbedoBuffer[linearIndex];
                Normal3D &accumNormal = plp.accumNormalBuffer[linearIndex];
                if (plp.numAccumFrames == 1) {
                    accumAlbedo = DiscretizedSpectrum::Zero();
                    accumNormal = Normal3D(0.0f, 0.0f, 0.0f);
                }
                TripletSpectrum whitePoint = createTripletSpectrum(SpectrumType::LightSource, ColorSpace::Rec709_D65,
                                                                   1, 1, 1);
                accumAlbedo += DiscretizedSpectrum(wls, exPayload.firstHitAlbedo * whitePoint.evaluate(wls) / plp.wavelengthProbability);
                accumNormal += exPayload.firstHitNormal;
                exPayloadPtr = nullptr;
            }

            if (rwPayload.pathLength >= MaxPathLength)
                rwPayload.terminate = true;
            if (rwPayload.terminate)
                break;

            rayOrg = woPayload.nextOrigin;
            rayDir = woPayload.nextDirection;
            roPayload.prevDirPDF = woPayload.dirPDF;
            roPayload.prevCosTerm = woPayload.cosTerm;
            roPayload.prevRevAreaPDF = woPayload.revAreaPDF;
            roPayload.prevSampledType = woPayload.sampledType;
        }
        plp.rngBuffer.write(launchIndex, rwPayload.rng);
        if (!rwPayload.contribution.allFinite()) {
            vlrprintf("Pass %u, (%u, %u): Not a finite value.\n", plp.numAccumFrames, launchIndex.x, launchIndex.y);
            return;
        }

        if (plp.numAccumFrames == 1)
            plp.accumBuffer[launchIndex].reset();
        plp.accumBuffer[launchIndex].add(wls, rwPayload.contribution);
    }



    CUDA_DEVICE_KERNEL void RT_CH_NAME(lvcbptEyePath)() {
        const auto hp = HitPointParameter::get();

        LVCBPTEyePathReadOnlyPayload* roPayload;
        LVCBPTEyePathWriteOnlyPayload* woPayload;
        LVCBPTEyePathReadWritePayload* rwPayload;
        LVCBPTEyePathExtraPayload* exPayload;
        LVCBPTEyePathPayloadSignature::get(&roPayload, &woPayload, &rwPayload, &exPayload);

        KernelRNG &rng = rwPayload->rng;
        WavelengthSamples wls = plp.commonWavelengthSamples;

        SurfacePoint surfPtE;
        float hypAreaPDF;
        calcSurfacePoint(hp, wls, &surfPtE, &hypAreaPDF);

        const SurfaceMaterialDescriptor matDescE = plp.materialDescriptorBuffer[hp.sbtr->geomInst.materialIndex];
        constexpr TransportMode transportModeE = TransportMode::Radiance;
        BSDF<transportModeE, BSDFTier::Bidirectional> bsdfE(matDescE, surfPtE, wls);
        EDF edf(matDescE, surfPtE, wls);

        if (exPayload) {
            exPayload->firstHitAlbedo = bsdfE.getBaseColor();
            exPayload->firstHitNormal = surfPtE.shadingFrame.z;
        }

        Vector3D dirOutE = -asVector3D(optixGetWorldRayDirection());
        Vector3D dirOutLocalE = surfPtE.shadingFrame.toLocal(dirOutE);

        Normal3D geomNormalLocalE = surfPtE.shadingFrame.toLocal(surfPtE.geometricNormal);
        BSDFQuery bsdfEQuery(dirOutLocalE, geomNormalLocalE, transportModeE, DirectionType::All(), wls);

        rwPayload->prevSumPowerProbDensities =
            rwPayload->prevTotalPowerProbDensity +
            pow2(roPayload->prevRevAreaPDF) * rwPayload->prevSumPowerProbDensities;
        rwPayload->prevTotalPowerProbDensity = rwPayload->totalPowerProbDensity;
        if (rwPayload->pathLength == 1) {
            // Ignore the strategy with zero eye vertices.
            rwPayload->prevSumPowerProbDensities = 0;
            // 
            float resCorrection = plp.imageSize.x * plp.imageSize.y;
            rwPayload->prevTotalPowerProbDensity *= pow2(resCorrection);
        }

        float lastDist2 = sqDistance(asPoint3D(optixGetWorldRayOrigin()), surfPtE.position);
        float probDensity = roPayload->prevDirPDF * absDot(dirOutLocalE, geomNormalLocalE) / lastDist2;
        //if (!vlr::isinf(rwPayload->totalPowerProbDensity) &&
        //    vlr::isinf(rwPayload->totalPowerProbDensity * pow2(probDensity))) {
        //    printf("EyePath: %g, %g\n", rwPayload->totalPowerProbDensity, pow2(probDensity));
        //}
        rwPayload->totalPowerProbDensity *= pow2(probDensity);

        // implicit light sampling
        SampledSpectrum spEmittance = edf.evaluateEmittance();
        if ((debugPathLength == 0 || rwPayload->pathLength == debugPathLength) &&
            spEmittance.hasNonZero()) {
            EDFQuery edfQuery(DirectionType::All(), wls);
            SampledSpectrum Le = spEmittance * edf.evaluate(edfQuery, dirOutLocalE);
            SampledSpectrum unweightedContribution = rwPayload->alpha * Le;

            const Instance &inst = plp.instBuffer[surfPtE.instIndex];
            float instProb = inst.lightGeomInstDistribution.integral() / plp.lightInstDist.integral();
            float geomInstProb = hp.sbtr->geomInst.importance / inst.lightGeomInstDistribution.integral();
            float forwardAreaDensityL = plp.numLightPaths * instProb * geomInstProb * hypAreaPDF;

            float backwardDirDensityE = edf.evaluatePDF(edfQuery, dirOutLocalE);
            float backwardAreaDensityE = backwardDirDensityE * roPayload->prevCosTerm / lastDist2;
            float partialDenomMisWeightE = rwPayload->prevTotalPowerProbDensity +
                pow2(backwardAreaDensityE) * rwPayload->prevSumPowerProbDensities; // extend light subpath, shorten eye subpath.

            float numMisWeight = rwPayload->totalPowerProbDensity;
            float denomMisWeight = numMisWeight;
            // extend light subpath, shorten eye subpath.
            denomMisWeight += pow2(forwardAreaDensityL) * partialDenomMisWeightE;

            float misWeight = 10 * numMisWeight / denomMisWeight;
            rwPayload->contribution += misWeight * unweightedContribution;
        }

        // Connect with a randomly chosen light vertex.
        if (bsdfE.hasNonDelta()) {
            uint32_t lightVertexIndex = vlr::min<uint32_t>(
                *plp.numLightVertices * rng.getFloat0cTo1o(),
                *plp.numLightVertices - 1);
            const LightPathVertex &vertex = plp.lightVertexCache[lightVertexIndex];
            float vertexProb = 1.0f / *plp.numLightVertices;

            SurfacePoint surfPtL;
            uint32_t matIndexL;
            {
                const GeometryInstance &geomInst = plp.geomInstBuffer[vertex.geomInstIndex];
                ProgSigDecodeHitPoint decodeHitPoint(geomInst.progDecodeHitPoint);
                decodeHitPoint(vertex.instIndex, vertex.geomInstIndex, vertex.primIndex,
                               vertex.u, vertex.v, &surfPtL);

                Normal3D localNormal = calcNode(geomInst.nodeNormal, Normal3D(0.0f, 0.0f, 1.0f), surfPtL, wls);
                applyBumpMapping(localNormal, &surfPtL);

                Vector3D newTangent = calcNode(geomInst.nodeTangent, surfPtL.shadingFrame.x, surfPtL, wls);
                modifyTangent(newTangent, &surfPtL);

                matIndexL = geomInst.materialIndex;
            }

            //printf("SurfPtL: p: (%g, %g, %g), frame: (%g, %g, %g), (%g, %g, %g), (%g, %g, %g), "
            //       "gn: (%g, %g, %g), %u - %u - %u - %g, %g, tc: %g, %g, inf: %u, point: %u\n",
            //       VLR3DPrint(surfPtL.position),
            //       VLR3DPrint(surfPtL.shadingFrame.x), VLR3DPrint(surfPtL.shadingFrame.y), VLR3DPrint(surfPtL.shadingFrame.z),
            //       VLR3DPrint(surfPtL.geometricNormal),
            //       surfPtL.instIndex, surfPtL.geomInstIndex, surfPtL.primIndex, surfPtL.u, surfPtL.v,
            //       surfPtL.texCoord.u, surfPtL.texCoord.v,
            //       surfPtL.atInfinity, surfPtL.isPoint);

            Vector3D conRayDir;
            float squaredConDist;
            float fractionalVisibility;
            if ((debugPathLength == 0 || (rwPayload->pathLength + vertex.pathLength + 1) == debugPathLength) &&
                testVisibility<LVCBPTRayType::Connection>(
                    surfPtE, surfPtL, wls, &conRayDir, &squaredConDist, &fractionalVisibility)) {
                float recSquaredConDist = 1.0f / squaredConDist;

                const SurfaceMaterialDescriptor matDescL = plp.materialDescriptorBuffer[matIndexL];
                constexpr TransportMode transportModeL = TransportMode::Importance;
                BSDF<transportModeL, BSDFTier::Bidirectional> bsdfL(matDescL, surfPtL, wls, vertex.pathLength == 0);
                Vector3D dirInLocalL = vertex.dirInLocal;
                Normal3D geomNormalLocalL = surfPtL.shadingFrame.toLocal(surfPtL.geometricNormal);
                BSDFQuery bsdfLQuery(dirInLocalL, geomNormalLocalL, transportModeL, DirectionType::All(), wls);

                Vector3D conRayDirLocalL = surfPtL.toLocal(-conRayDir);
                Vector3D conRayDirLocalE = surfPtE.toLocal(conRayDir);

                float cosL = absDot(conRayDirLocalL, geomNormalLocalL);
                float cosE = absDot(conRayDirLocalE, geomNormalLocalE);
                float G = cosL * cosE * recSquaredConDist;

                SampledSpectrum backwardFsL;
                SampledSpectrum forwardFsL = bsdfL.evaluate(bsdfLQuery, conRayDirLocalL, &backwardFsL);
                float backwardDirDensityL;
                float forwardDirDensityL = bsdfL.evaluatePDF(bsdfLQuery, conRayDirLocalL, &backwardDirDensityL);
                float forwardAreaDensityL = forwardDirDensityL * cosE * recSquaredConDist;
                float backwardAreaDensityL = backwardDirDensityL * vertex.backwardConversionFactor;
                float partialDenomMisWeightL = vertex.prevTotalPowerProbDensity +
                    pow2(backwardAreaDensityL) * vertex.prevSumPowerProbDensities; // extend eye subpath, shorten light subpath.

                SampledSpectrum backwardFsE;
                SampledSpectrum forwardFsE = bsdfE.evaluate(bsdfEQuery, conRayDirLocalE, &backwardFsE);
                float backwardDirDensityE;
                float forwardDirDensityE = bsdfE.evaluatePDF(bsdfEQuery, conRayDirLocalE, &backwardDirDensityE);
                float forwardAreaDensityE = forwardDirDensityE * cosL * recSquaredConDist;
                float backwardAreaDensityE = backwardDirDensityE * roPayload->prevCosTerm / lastDist2;
                float partialDenomMisWeightE = rwPayload->prevTotalPowerProbDensity +
                    pow2(backwardAreaDensityE) * rwPayload->prevSumPowerProbDensities; // extend light subpath, shorten eye subpath.

                float scalarTerm = G * fractionalVisibility /
                    (vertexProb * plp.wavelengthProbability);
                if (vertex.wlSelected || rwPayload->singleIsSelected)
                    scalarTerm *= SampledSpectrum::NumComponents();
                SampledSpectrum conTerm = forwardFsL * scalarTerm * forwardFsE;
                SampledSpectrum unweightedContribution = vertex.flux * conTerm * rwPayload->alpha;

                float numMisWeight = rwPayload->totalPowerProbDensity * vertex.totalPowerProbDensity;
                float denomMisWeight = numMisWeight;
                // extend eye subpath, shorten light subpath.
                denomMisWeight += rwPayload->totalPowerProbDensity * pow2(forwardAreaDensityE) * partialDenomMisWeightL;
                // extend light subpath, shorten eye subpath.
                denomMisWeight += vertex.totalPowerProbDensity * pow2(forwardAreaDensityL) * partialDenomMisWeightE;

                float misWeight = 10 * numMisWeight / denomMisWeight;
                rwPayload->contribution += misWeight * unweightedContribution;

                //if (!vlr::isfinite(misWeight) || !unweightedContribution.allFinite()) {
                //    printf("%g (%g, %g), (%g, %g), (%g, %g, %g, %g), (%g, %g, %g)\n",
                //           misWeight, numMisWeight, denomMisWeight,
                //           rwPayload->totalPowerProbDensity, vertex.totalPowerProbDensity,
                //           probDensity, roPayload->prevDirPDF, absDot(dirOutLocalE, geomNormalLocalE), lastDist2,
                //           unweightedContribution.r, unweightedContribution.g, unweightedContribution.b);
                //}
            }
        }

        BSDFSample sample(rng.getFloat0cTo1o(), rng.getFloat0cTo1o(), rng.getFloat0cTo1o());
        BSDFQueryResult fsResult;
        BSDFQueryReverseResult fsRevResult;
        SampledSpectrum fs = bsdfE.sample(bsdfEQuery, sample, &fsResult, &fsRevResult);
        if (fs == SampledSpectrum::Zero() || fsResult.dirPDF == 0.0f)
            return;
        if (fsResult.sampledType.isDispersive() && !wls.singleIsSelected())
            rwPayload->singleIsSelected = true;

        float cosTerm = dot(fsResult.dirLocal, geomNormalLocalE);
        SampledSpectrum throughput = fs * (std::fabs(cosTerm) / fsResult.dirPDF);
        rwPayload->alpha *= throughput;

        // Russian roulette
        float continueProb = std::fmin(throughput.importance(wls.selectedLambdaIndex()), 1.0f);
        if (rng.getFloat0cTo1o() >= continueProb)
            return;
        rwPayload->alpha /= continueProb;
        rwPayload->terminate = false;

        Vector3D dirInE = surfPtE.fromLocal(fsResult.dirLocal);
        woPayload->nextOrigin = offsetRayOrigin(surfPtE.position, cosTerm > 0.0f ? surfPtE.geometricNormal : -surfPtE.geometricNormal);
        woPayload->nextDirection = dirInE;
        woPayload->dirPDF = fsResult.dirPDF;
        woPayload->cosTerm = std::fabs(cosTerm);
        woPayload->revAreaPDF = fsRevResult.dirPDF * roPayload->prevCosTerm / lastDist2;
        woPayload->sampledType = fsResult.sampledType;
    }



    CUDA_DEVICE_KERNEL void RT_MS_NAME(lvcbptEyePath)() {
        LVCBPTEyePathReadOnlyPayload* roPayload;
        LVCBPTEyePathReadWritePayload* rwPayload;
        LVCBPTEyePathExtraPayload* exPayload;
        LVCBPTEyePathPayloadSignature::get(&roPayload, nullptr, &rwPayload, &exPayload);

        if (exPayload) {
            exPayload->firstHitAlbedo = SampledSpectrum::Zero();
            exPayload->firstHitNormal = Normal3D(0.0f, 0.0f, 0.0f);
        }
    }
}
